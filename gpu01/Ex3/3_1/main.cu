#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "chTimer.h"


int main() {
    const int minSize = 1024;  // 1 KB
    const int maxSize = 1024 * 1024 * 1024;  // 1 GB
    const int numIterations = 500;
    chTimerTimestamp start,stop;

    for (long size = minSize; size <= maxSize; size *= 2) {
        if(size >= maxSize){ size = maxSize; }
        float* h_data = (float*)malloc((size/4) * sizeof(float));
        float* d_data;
        hipMalloc((void**)&d_data, (size/4) * sizeof(float));

        // Measure host-to-device bandwidth
        chTimerGetTime( &start );
        for (int iter = 0; iter < numIterations; iter++) {
            hipMemcpy(d_data, h_data, (size/4) * sizeof(float), hipMemcpyHostToDevice);
            hipDeviceSynchronize();
        }
        chTimerGetTime( &stop );
        double elapsedSeconds = chTimerElapsedTime( &start, &stop );
        double hostToDeviceBandwidth = ((size/4) * sizeof(float) * numIterations) / (elapsedSeconds * 1e9); // GB/s

        // Measure device-to-host bandwidth
        chTimerGetTime( &start );
        for (int iter = 0; iter < numIterations; iter++) {
            hipMemcpy(h_data, d_data, (size/4) * sizeof(float), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
        }
        chTimerGetTime( &stop );
        elapsedSeconds = chTimerElapsedTime( &start, &stop );
        double deviceToHostBandwidth = ((size/4) * sizeof(float) * numIterations) / (elapsedSeconds * 1e9); // GB/s

        // Measure device-to-device bandwith
        float* d_data_2;
        hipMalloc((void**)&d_data_2, (size/4) * sizeof(float));
        chTimerGetTime( &start );
        for (int iter = 0; iter < numIterations; iter++) {
            hipMemcpy(d_data_2, d_data, (size/4) * sizeof(float), hipMemcpyDeviceToDevice);
            hipDeviceSynchronize();
        }
        chTimerGetTime( &stop );
        elapsedSeconds = chTimerElapsedTime( &start, &stop );
        double deviceToDeviceBandwidth = ((size/4) * sizeof(float) * numIterations) / (elapsedSeconds * 1e9); // GB/s

        printf("Data Size: %d KB\tHost-to-Device Bandwidth: %lf GB/s\tDevice-to-Host Bandwidth: %lf GB/s\tDevice-to-Device Bandwidth: %lf GB/s\n", size / 1024, hostToDeviceBandwidth, deviceToHostBandwidth, deviceToDeviceBandwidth);

        // Clean up
        hipFree(d_data);
        hipFree(d_data_2);
        free(h_data);
        hipDeviceSynchronize();
    }

    return 0;
}