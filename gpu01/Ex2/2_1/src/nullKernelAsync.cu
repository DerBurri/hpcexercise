#include "hip/hip_runtime.h"
/*
 *
 * nullKernelAsync.cu
 *
 * Microbenchmark for throughput of asynchronous kernel launch.
 *
 * Build with: nvcc -I ../chLib <options> nullKernelAsync.cu
 * Requires: No minimum SM requirement.
 *
 * Copyright (c) 2011-2012, Archaea Software, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions 
 * are met: 
 *
 * 1. Redistributions of source code must retain the above copyright 
 *    notice, this list of conditions and the following disclaimer. 
 * 2. Redistributions in binary form must reproduce the above copyright 
 *    notice, this list of conditions and the following disclaimer in 
 *    the documentation and/or other materials provided with the 
 *    distribution. 
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS 
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE 
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, 
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER 
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT 
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN 
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE 
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include <stdio.h>

#include "chTimer.h"

__global__ void NullKernel()
{
}

int main()
{
    const int cIterations = 10000;
    const int meas_iterations = 10;

    chTimerTimestamp start, stop;


    for(int tpb = 1; tpb <= 1024; tpb+=100) {
	for(int nb = 1; nb < 16384; nb+=100) {
	    for(int j = 0; j < meas_iterations; j++) {

		printf( "SYNC - NB: %d, TPB: %d\t-> ", nb, tpb); fflush( stdout );

		chTimerGetTime( &start );

		for ( int i = 0; i < cIterations; i++ ) {
		    // numBlocks, threadsPerBlock
		    NullKernel <<< nb, tpb >>>();
		    hipDeviceSynchronize();
		}

		// Wait for all previous threads to complete
		// hipDeviceSynchronize();

		chTimerGetTime( &stop );

		{
		    double microseconds = 1e6*chTimerElapsedTime( &start, &stop );
		    double usPerLaunch = microseconds / (float) cIterations;

		    printf( "%.2f us\n", usPerLaunch );
		}
	    }
	}
    }

    return 0;
}
