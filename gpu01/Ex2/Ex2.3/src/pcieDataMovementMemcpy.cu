#include<stdio.h>
#include<chTimer.h>

const int cIterations = 1000000;

chTimerTimestamp start, stop;

void test(size_t size, int* dintArray, int* hintArray, hipMemcpyKind direction) {

  //Host 2 Device
  for (size_t i = 0; i < cIterations; i++)
  {
    if (direction == hipMemcpyHostToDevice) {
    hipMemcpy(dintArray, hintArray, size*sizeof(int),direction);
    hipDeviceSynchronize();
    }
    else if(direction == hipMemcpyDeviceToHost) {
    hipMemcpy(hintArray,dintArray,size*sizeof(int),direction);
    cudaDeviceSynchrqonize();
    }
  }

  
}

int main()
{   
  //Allocate Memory on Host and Device
  const int allocatedMemorySize = 1*1024*1024*1024; //1GB
  int *hintArray, *dintArray;

  printf("Measuring transfeir from Host to Device with malloc allocated pageable memory");
  
  for (size_t i = 0; i < 10; i++)
  {
    hintArray = (int*)malloc(allocatedMemorySize*sizeof(int));
    hipMalloc(&dintArray,allocatedMemorySize*sizeof(int));
    
    chTimerGetTime( &start );
    test(allocatedMemorySize,dintArray,hintArray,hipMemcpyHostToDevice);
    chTimerGetTime(&stop);
    free(hintArray);
    hipFree(dintArray);
    {
    double microseconds = 1e6*chTimerElapsedTime( &start, &stop );
    double usPerLaunch = microseconds / (float) cIterations;
    printf( "%.4f us\n", usPerLaunch );
  }

  }


  printf("Measuring transfer from Device to Host with malloc allocated pageable memory");

  for (size_t i = 0; i< 10; i++)
  {
    hintArray = (int*)malloc(allocatedMemorySize*sizeof(int));
    hipMalloc(&dintArray,allocatedMemorySize*sizeof(int));
    chTimerGetTime( &start );
    test(allocatedMemorySize,dintArray,hintArray,hipMemcpyDeviceToHost);
    chTimerGetTime(&stop);
    free(hintArray);
    hipFree(dintArray);
    {
    double microseconds = 1e6*chTimerElapsedTime( &start, &stop );
    double usPerLaunch = microseconds / (float) cIterations;
    printf( "%.4f us\n", usPerLaunch );
  }

  }
//Test with hipMalloc 

  printf("Measuring transfer from Host to Device with hipHostMalloc pinned memory");
  for (size_t i = 0; i < 10; i++)
  {
    hipHostMalloc(&hintArray,allocatedMemorySize);
    hipMalloc(&dintArray,allocatedMemorySize*sizeof(int));
    chTimerGetTime( &start );
    test(allocatedMemorySize,dintArray,hintArray,hipMemcpyHostToDevice);
    chTimerGetTime(&stop);
    hipFree(hintArray);
    hipFree(dintArray);
    {
    double microseconds = 1e6*chTimerElapsedTime( &start, &stop );
    double usPerLaunch = microseconds / (float) cIterations;
    printf( "%.4f us\n", usPerLaunch );
  }
  }
    printf("Measuring transfer from Device to Host with hipHostMalloc pinned memory");

  for (size_t i = 0; i < 10; i++)
  {
    hipHostMalloc(&hintArray,allocatedMemorySize);
    hipMalloc(&dintArray,allocatedMemorySize*sizeof(int));
    test(allocatedMemorySize,dintArray,hintArray,hipMemcpyDeviceToHost);
    chTimerGetTime(&stop);
    hipFree(hintArray);
    hipFree(dintArray);
    {
    double microseconds = 1e6*chTimerElapsedTime( &start, &stop );
    double usPerLaunch = microseconds / (float) cIterations;
    printf( "%.4f us\n", usPerLaunch );
  }
  }
//

return 0;
}

