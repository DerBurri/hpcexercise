/**************************************************************************************************
 *
 *       Computer Engineering Group, Heidelberg University - GPU Computing Exercise 06
 *
 *                 Gruppe : 01
 *
 *                   File : kernel.cu
 *
 *                Purpose : Reduction
 *
 **************************************************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
//
// Reduction_Kernel
//
__global__ void reduction_Kernel(int numElements, float* dataIn, float* dataOut)
{
	int elementId = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = numElements / (blockDim.x * gridDim.x);
    int halfstride = stride/2;
    __shared__ float value[1];
    value[0] = 0;

	if (elementId < numElements)
	{

        if(gridDim.x > 1)
        {
            atomicAdd_system(value, dataIn[elementId * stride]);
            __syncthreads();
            atomicAdd_system(value, dataIn[(elementId * stride) + halfstride]);
            __syncthreads();

            if(threadIdx.x == 0)
            {
                dataIn[elementId * stride] = value[0];
            }
        }
        else
        {
           
             __syncthreads();
            atomicAdd_system(value, dataIn[elementId * stride]);
            __syncthreads();
            *dataOut = value[0];
        }
	}
}


void reduction_Kernel_Wrapper(dim3 gridSize, dim3 blockSize, int numElements, float* dataIn, float* dataOut)
{
	reduction_Kernel<<< gridSize, blockSize>>>(numElements, dataIn, dataOut);
}
