/**************************************************************************************************
 *
 *       Computer Engineering Group, Heidelberg University - GPU Computing Exercise 06
 *
 *                 Gruppe : TODO
 *
 *                   File : kernel.cu
 *
 *                Purpose : Reduction
 *
 **************************************************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
//
// Reduction_Kernel
//
__global__ void reduction_Kernel(int numElements, float* dataIn, float* dataOut)
{
	int elementId = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = numElements / (blockDim.x * gridDim.x);
    int halfstride = stride/2;
    float value = 0;
    // printf("elementId %d, stride %d, halfstride %d\n");
	if (elementId < numElements)
	{
        value = dataIn[elementId * stride] + dataIn[(elementId * stride) + halfstride];
        __syncthreads();
        dataIn[elementId * stride] = value;
        printf("%f ", dataIn[elementId * stride]);
	}
}


void reduction_Kernel_Wrapper(dim3 gridSize, dim3 blockSize, int numElements, float* dataIn, float* dataOut)
{
	reduction_Kernel<<< gridSize, blockSize>>>(numElements, dataIn, dataOut);
}
