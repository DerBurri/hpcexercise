
#include <hip/hip_runtime.h>
/**************************************************************************************************
 *
 *       Computer Engineering Group, Heidelberg University - GPU Computing Exercise 06
 *
 *                 Gruppe : TODO
 *
 *                   File : kernel.cu
 *
 *                Purpose : Reduction
 *
 **************************************************************************************************/

//
// Reduction_Kernel
//
__global__ void
reduction_Kernel(int numElements, float* dataIn, float* dataOut)
{
	int elementId = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (elementId < numElements)
	{
		/*TODO Kernel Code*/
	}
}

void reduction_Kernel_Wrapper(dim3 gridSize, dim3 blockSize, int numElements, float* dataIn, float* dataOut) {
	reduction_Kernel<<< gridSize, blockSize>>>(numElements, dataIn, dataOut);
}
