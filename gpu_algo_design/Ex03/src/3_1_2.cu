#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void __lane_min_int(hiprandState *state) {
  int idx = threadIdx.x;

  // Initialize cuRAND
  hiprand_init(1234, idx, 0, &state[idx]);

  // Generate a random positive integer
  int local_value = static_cast<int>(hiprand_uniform(&state[idx]) *
                                     1000);  // Random positive integer
  printf("Thread %d has value %d\n", idx, local_value);

  int min_value = local_value;
  int min_index = idx;

  // Use __reduce_min_sync to find the minimum value in the warp
  // Disabled Code
  //int warp_min_value = __reduce_min_sync(0xffffffff, local_value);
  //int warp_min_index = __reduce_min_sync(0xffffffff, idx);

  // // shuffle down to find the min value and its index
  // for (int i = 16; i > 0; i /= 2) {
  //   //int temp_value = __shfl_down_sync(0xffffffff, min_value, i);
  //   //int temp_index = __shfl_down_sync(0xffffffff, min_index, i);
  //   if (temp_value < min_value) {
  //     min_value = temp_value;
  //     min_index = temp_index;
  //   }
  // }
  int rotated_value = __shfl_sync(0xFFFFFFFF, local_value, (threadIdx.x -1 ) & (32 - 1));

  min_index = __shfl_sync(0xffffffff, min_index, 0);
  printf("Thread %d, New Value %d Old Value %d\n",idx, rotated_value, local_value);

}

int main() {
  printf("test\n");
  // Allocate space for cuRAND state
  hiprandState *d_state;
  hipMalloc(&d_state, 32 * sizeof(hiprandState));

  // Launch kernel
  __lane_min_int<<<1, 32>>>(d_state);
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("CUDA Error: %s\n", hipGetErrorString(err));
  }
  hipDeviceSynchronize();

  // Free cuRAND state
  hipFree(d_state);

  return 0;
}