#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void __lane_min_int(hiprandState *state) {
  __shared__ int idx_min;
  int idx = threadIdx.x;

  // Initialize cuRAND
  hiprand_init(1234, idx, 0, &state[idx]);

  // Generate a random positive integer
  int local_value = static_cast<int>(hiprand_uniform(&state[idx]) *
                                     1000);  // Random positive integer
  printf("Thread %d has value %d\n", idx, local_value);

  int min_value = local_value;
  int min_index = idx;

  // Use __reduce_min_sync to find the minimum value in the warp
  int warp_min_value = __reduce_min_sync(0xffffffff, local_value);
  int warp_min_index = __reduce_min_sync(0xffffffff, idx);

  // shuffle down to find the min value and its index
  for (int i = 16; i > 0; i /= 2) {
    int temp_value = __shfl_down_sync(0xffffffff, min_value, i);
    int temp_index = __shfl_down_sync(0xffffffff, min_index, i);
    if (temp_value < min_value) {
      min_value = temp_value;
      min_index = temp_index;
    }
  }

  if (threadIdx.x == 0) idx_min = min_index;
  if (threadIdx.x == 0) {
    printf("Lane index of the min value in warp using __shfl_down_sync is %d\n",
           idx_min);
    printf("Min value in warp using __shfl_down_sync is %d\n", min_value);
  }
}

int main() {
  printf("test\n");
  // Allocate space for cuRAND state
  hiprandState *d_state;
  hipMalloc(&d_state, 32 * sizeof(hiprandState));

  // Launch kernel
  __lane_min_int<<<1, 32>>>(d_state);
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("CUDA Error: %s\n", hipGetErrorString(err));
  }
  hipDeviceSynchronize();

  // Free cuRAND state
  hipFree(d_state);

  return 0;
}