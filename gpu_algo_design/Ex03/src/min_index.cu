#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void __warp_min_idx(hiprandState *state) {
  int idx_min;
  int idx = threadIdx.x;
  int min_value;
  int min_index = idx;

  // Initialize cuRAND
  int seed = 1234;
  hiprand_init(seed, idx, 0, &state[idx]);

  // Generate a random positive integer
  int local_value = static_cast<int>(hiprand_uniform(&state[idx]) *
                                     1000);  // Random positive integer
  printf("Thread %d has value %d\n", idx, local_value);
  min_value = local_value;

  // shuffle down to find the min value and its index
  for (int i = 16; i > 0; i /= 2) {
    int temp_value = __shfl_down_sync(0xffffffff, min_value, i);
    int temp_index = __shfl_down_sync(0xffffffff, min_index, i);
    if (temp_value < min_value) {
      min_value = temp_value;
      min_index = temp_index;
    }
  }

  min_index = __shfl_sync(0xffffffff, min_index, 0);
  if (threadIdx.x == 0) {
    printf("Lane index of the min value in warp is %d\n",
           idx_min);
    printf("Min value in warp is %d\n", min_value);
  }
}

int main() {
  printf("test\n");
  // Allocate space for cuRAND state
  hiprandState *d_state;
  hipMalloc(&d_state, 32 * sizeof(hiprandState));

  // Launch kernel
  __warp_min_idx<<<1, 32>>>(d_state);
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("CUDA Error: %s\n", hipGetErrorString(err));
  }
  hipDeviceSynchronize();

  // Free cuRAND state
  hipFree(d_state);

  return 0;
}