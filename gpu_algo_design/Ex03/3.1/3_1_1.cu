#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <stdlib.h>

#define SHFL_DELTA 4
__global__ void __shfl_up_wrap(hiprandState *state) {
  int idx = threadIdx.x;

  // Initialize cuRAND
  hiprand_init(1234, idx, 0, &state[idx]);

  // Generate a random positive integer
  int local_value = static_cast<int>(hiprand_uniform(&state[idx]) *
                                     1000);  // Random positive integer

  int rotated_value = __shfl_sync(0xFFFFFFFF, local_value, (threadIdx.x - SHFL_DELTA ) & (32 - 1));
  printf("Thread %d, New Value %d Old Value %d\n",idx, rotated_value, local_value);

}

int main() {
  printf("test\n");
  // Allocate space for cuRAND state
  hiprandState *d_state;
  hipMalloc(&d_state, 32 * sizeof(hiprandState));

  // Launch kernel
  __shfl_up_wrap<<<1, 32>>>(d_state);
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("CUDA Error: %s\n", hipGetErrorString(err));
  }
  hipDeviceSynchronize();

  // Free cuRAND state
  hipFree(d_state);

  return 0;
}