#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This is a simple test program to measure the memcopy bandwidth of the GPU.
 * It can measure device to device copy bandwidth, host to device copy bandwidth
 * for pageable and pinned memory, and device to host copy bandwidth for
 * pageable and pinned memory.
 *
 * Usage:
 * ./bandwidthTest [option]...
 */

// CUDA runtime
#include <hip/hip_runtime.h>

// includes
#include <hip/hip_runtime_api.h>      // helper functions for CUDA error checking and initialization
#include <helper_functions.h> // helper for shared functions common to CUDA Samples

#include <hip/hip_runtime.h>

#include <cassert>
#include <iostream>
#include <memory>

static const char *sSDKsample = "CUDA Bandwidth Test";

// defines, project
#define MEMCOPY_ITERATIONS 1
#define DEFAULT_SIZE (32 * (1e6))     // 32 M
#define DEFAULT_INCREMENT (4 * (1e6)) // 4 M
#define DEFAULT_BYTES_PER_INSTRUCTION 4
#define CACHE_CLEAR_SIZE (16 * (1e6)) // 16 M

// shmoo mode defines
#define SHMOO_MEMSIZE_MAX (300 * (1e6))      // 64 M
#define SHMOO_MEMSIZE_START (1e3)           // 1 KB
#define SHMOO_INCREMENT_1KB (1e3)           // 1 KB
#define SHMOO_INCREMENT_2KB (2 * 1e3)       // 2 KB
#define SHMOO_INCREMENT_10KB (10 * (1e3))   // 10KB
#define SHMOO_INCREMENT_100KB (100 * (1e3)) // 100 KB
#define SHMOO_INCREMENT_1MB (1e6)           // 1 MB
#define SHMOO_INCREMENT_2MB (2 * 1e6)       // 2 MB
#define SHMOO_INCREMENT_4MB (4 * 1e6)       // 4 MB
#define SHMOO_INCREMENT_8MB (8 * 1e6)       // 8 MB
#define SHMOO_INCREMENT_13MB (13 * 1e6)       // 13 MB
#define SHMOO_INCREMENT_25MB (25 * 1e6)       // 25 MB
#define SHMOO_LIMIT_20KB (20 * (1e3))       // 20 KB
#define SHMOO_LIMIT_50KB (50 * (1e3))       // 50 KB
#define SHMOO_LIMIT_100KB (100 * (1e3))     // 100 KB
#define SHMOO_LIMIT_1MB (1e6)               // 1 MB
#define SHMOO_LIMIT_16MB (16 * 1e6)         // 16 MB
#define SHMOO_LIMIT_32MB (32 * 1e6)         // 32 MB
#define SHMOO_LIMIT_64MB (64 * 1e6)         // 64 MB
#define SHMOO_LIMIT_128MB (128 * 1e6)         // 128 MB
#define SHMOO_LIMIT_256MB (256 * 1e6)         // 256 MB

// CPU cache flush
#define FLUSH_SIZE (256 * 1024 * 1024)
char *flush_buf;

// enums, project
enum testMode
{
  QUICK_MODE,
  RANGE_MODE,
  SHMOO_MODE
};
enum memcpyKind
{
  DEVICE_TO_HOST,
  HOST_TO_DEVICE,
  DEVICE_TO_DEVICE
};
enum printMode
{
  USER_READABLE,
  CSV
};
enum memoryMode
{
  PINNED,
  PAGEABLE
};
enum kernelMode
{
  DEFAULT,
  KERNEL_MODE,
  KERNEL2_MODE,
  KERNEL3_MODE
};

const char *sMemoryCopyKind[] = {"Device to Host", "Host to Device",
                                 "Device to Device", NULL};

const char *sMemoryMode[] = {"PINNED", "PAGEABLE", NULL};

// if true, use CPU based timing for everything
static bool bDontUseGPUTiming;

int *pArgc = NULL;
char **pArgv = NULL;

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
int runTest(const int argc, const char **argv);
void testBandwidth(unsigned int start, unsigned int end, unsigned int increment,
                   testMode mode, memcpyKind kind, printMode printmode,
                   memoryMode memMode, int startDevice, int endDevice, bool wc, kernelMode kernelMode, int bytes_per_inst, char* dtypeStr);
void testBandwidthQuick(unsigned int size, memcpyKind kind, printMode printmode,
                        memoryMode memMode, int startDevice, int endDevice,
                        bool wc, kernelMode mode, int bytes_per_inst);
void testBandwidthRange(unsigned int start, unsigned int end,
                        unsigned int increment, memcpyKind kind,
                        printMode printmode, memoryMode memMode,
                        int startDevice, int endDevice, bool wc, kernelMode mode, int bytes_per_inst);
template <typename T>
void testBandwidthShmoo(memcpyKind kind, printMode printmode,
                        memoryMode memMode, int startDevice, int endDevice,
                        bool wc, kernelMode mode, int bytes_per_inst, char* dtypeStr);
template <typename T>
float testDeviceToHostTransfer(unsigned int memSize, memoryMode memMode,
                               bool wc, kernelMode mode, int bytes_per_inst);
template <typename T>
float testHostToDeviceTransfer(unsigned int memSize, memoryMode memMode,
                               bool wc, kernelMode mode, int bytes_per_inst);
template <typename T>
float testDeviceToDeviceTransfer(unsigned int memSize, kernelMode mode, int bytes_per_inst);
void printResultsReadable(unsigned int *memSizes, double *bandwidths,
                          unsigned int count, memcpyKind kind,
                          memoryMode memMode, int iNumDevs, bool wc);
void printResultsCSV(unsigned int *memSizes, double *bandwidths,
                     unsigned int count, memcpyKind kind, memoryMode memMode,
                     int iNumDevs, bool wc, kernelMode mode, int bytes_per_inst, std::string dtypeStr);
void printHelp(void);

template <typename KernelType>
void calculateKernelConfig(int numElements, dim3 &grid, dim3 &block, KernelType kernel)
{
  int minGridSize, blockSize;
  hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, kernel, 0, 0);

  grid.x = (numElements + blockSize - 1) / blockSize;
  block.x = blockSize;
  // Calculated Kernel Launch Configuration
  //printf("Kernel Launch Configuration ");
  //printf("Grid Size: %d, Block Size: %d", grid.x, block.x);
}

////////////////////////////////////////////////////////////////////////////////
// Kernels
////////////////////////////////////////////////////////////////////////////////

template <typename T>
__global__ void copyKernel(const T *in, T *out, size_t num_bytes)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < num_bytes)
  {
    out[i] = in[i];
  }
}

template <typename T>
__global__ void copyKernel2(const T *__restrict__ in,
                            T *__restrict__ out,
                            size_t num_bytes, int bytes_per_inst)
{
  unsigned int total_threads = gridDim.x * blockDim.x;

  // num_bytes = number of all elements to copy
  unsigned int inst_per_thread = num_bytes / bytes_per_inst / total_threads;
  if (inst_per_thread == 0)
    inst_per_thread = 1;
  unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
  //printf("thread_id %d, inst_per_thread %d\n", threadIdx.x, inst_per_thread);
  if (num_bytes > idx * (bytes_per_inst * inst_per_thread))
  {
    for (unsigned int element_idx = idx * (bytes_per_inst * inst_per_thread);
         element_idx < (idx + 1) * (bytes_per_inst * inst_per_thread);
         element_idx += bytes_per_inst)
    {
      auto lower_border = idx * (bytes_per_inst * inst_per_thread);
      auto upper_border = (idx + 1) * (bytes_per_inst * inst_per_thread);
      //printf("thread_id %d, element_idx: %d, lower_border: %d, upper_border: %d\n", threadIdx.x, element_idx, lower_border, upper_border);
      switch (bytes_per_inst)
      {
      case 4:
        if (element_idx * 4 < num_bytes)
        {
          reinterpret_cast<int *>(out)[idx] =
              reinterpret_cast<const int *>(in)[idx];
        }
        break;
      case 8:
        if (element_idx * 8 < num_bytes)
        {
          reinterpret_cast<int2 *>(out)[idx] =
              reinterpret_cast<const int2 *>(in)[idx];
        }
        break;
      case 16:
        if (element_idx * 16 < num_bytes)
        {
          reinterpret_cast<int4 *>(out)[idx] =
              reinterpret_cast<const int4 *>(in)[idx];
        }
        break;
      default:
        hipError_t error = hipErrorInvalidValue;
        break;
      }
    }
  }
  // Handle remaining bytes if any
  int remaining_bytes = num_bytes % bytes_per_inst;
  int remaining_start_idx = num_bytes - remaining_bytes;

  if (idx == 0 && remaining_bytes > 0)
  {
    for (int i = 0; i < remaining_bytes; ++i)
    {
      out[remaining_start_idx + i] = in[remaining_start_idx + i];
    }
  }
}

template <typename T, class Functor>
__global__ void transformKernel(const T *in, T *out, size_t num_elements, Functor f)
{
  const int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < num_elements)
  {
    T in_value;

    memcpy(&in_value, &in[i], sizeof(T));

    // Apply the functir
    T out_value = f(in_value);

    memcpy(&out[i], &out_value, sizeof(T));
  }
}

// Example Functor
struct MultiplyByTwo {
  __host__ __device__ float operator()(float x) const { return x * 2; }
};

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
  pArgc = &argc;
  pArgv = argv;

  flush_buf = (char *)malloc(FLUSH_SIZE);

  // set logfile name and start logs
  // printf("[%s] - Starting...\n", sSDKsample);

  int iRetVal = runTest(argc, (const char **)argv);

  if (iRetVal < 0)
  {
    checkCudaErrors(hipSetDevice(0));
  }

  // finish
  // printf("%s\n", (iRetVal == 0) ? "Result = PASS" : "Result = FAIL");

  // printf(
  //     "\nNOTE: The CUDA Samples are not meant for performance measurements. "
  //     "Results may vary when GPU Boost is enabled.\n");

  free(flush_buf);

  exit((iRetVal == 0) ? EXIT_SUCCESS : EXIT_FAILURE);
}

///////////////////////////////////////////////////////////////////////////////
// Parse args, run the appropriate tests
///////////////////////////////////////////////////////////////////////////////
int runTest(const int argc, const char **argv)
{
  int start = DEFAULT_SIZE;
  int end = DEFAULT_SIZE;
  int startDevice = 0;
  int endDevice = 0;
  int increment = DEFAULT_INCREMENT;
  int bytes_per_inst = DEFAULT_BYTES_PER_INSTRUCTION;
  testMode mode = QUICK_MODE;
  kernelMode kernel = DEFAULT;
  bool htod = false;
  bool dtoh = false;
  bool dtod = false;
  bool wc = false;
  char *modeStr;
  char *kernelStr;
  char *dtypeStr;
  char *device = NULL;
  printMode printmode = USER_READABLE;
  char *memModeStr = NULL;
  memoryMode memMode = PINNED;

  // process command line args
  if (checkCmdLineFlag(argc, argv, "help"))
  {
    printHelp();
    return 0;
  }

  if (checkCmdLineFlag(argc, argv, "csv"))
  {
    printmode = CSV;
  }

  if (getCmdLineArgumentString(argc, argv, "memory", &memModeStr))
  {
    if (strcmp(memModeStr, "pageable") == 0)
    {
      memMode = PAGEABLE;
    }
    else if (strcmp(memModeStr, "pinned") == 0)
    {
      memMode = PINNED;
    }
    else
    {
      printf("Invalid memory mode - valid modes are pageable or pinned\n");
      printf("See --help for more information\n");
      return -1000;
    }
  }
  else
  {
    // default - pinned memory
    memMode = PINNED;
  }

  if (getCmdLineArgumentString(argc, argv, "device", &device))
  {
    int deviceCount;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);

    if (error_id != hipSuccess)
    {
      printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error_id,
             hipGetErrorString(error_id));
      exit(EXIT_FAILURE);
    }

    if (deviceCount == 0)
    {
      printf("!!!!!No devices found!!!!!\n");
      return -2000;
    }

    if (strcmp(device, "all") == 0)
    {
      printf(
          "\n!!!!!Cumulative Bandwidth to be computed from all the devices "
          "!!!!!!\n\n");
      startDevice = 0;
      endDevice = deviceCount - 1;
    }
    else
    {
      startDevice = endDevice = atoi(device);

      if (startDevice >= deviceCount || startDevice < 0)
      {
        printf(
            "\n!!!!!Invalid GPU number %d given hence default gpu %d will be "
            "used !!!!!\n",
            startDevice, 0);
        startDevice = endDevice = 0;
      }
    }
  }

  //printf("Running on...\n\n");

  for (int currentDevice = startDevice; currentDevice <= endDevice;
       currentDevice++)
  {
    hipDeviceProp_t deviceProp;
    hipError_t error_id = hipGetDeviceProperties(&deviceProp, currentDevice);

    if (error_id == hipSuccess)
    {
      //printf(" Device %d: %s\n", currentDevice, deviceProp.name);

      if (deviceProp.computeMode == hipComputeModeProhibited)
      {
        fprintf(stderr,
                "Error: device is running in <Compute Mode Prohibited>, no "
                "threads can use ::hipSetDevice().\n");
        checkCudaErrors(hipSetDevice(currentDevice));

        exit(EXIT_FAILURE);
      }
    }
    else
    {
      printf("hipGetDeviceProperties returned %d\n-> %s\n", (int)error_id,
             hipGetErrorString(error_id));
      checkCudaErrors(hipSetDevice(currentDevice));

      exit(EXIT_FAILURE);
    }
  }

  if (getCmdLineArgumentString(argc, argv, "mode", &modeStr))
  {
    // figure out the mode
    if (strcmp(modeStr, "quick") == 0)
    {
      //printf(" Quick Mode\n\n");
      mode = QUICK_MODE;
    }
    else if (strcmp(modeStr, "shmoo") == 0)
    {
      //printf(" Shmoo Mode\n\n");
      mode = SHMOO_MODE;
    }
    else if (strcmp(modeStr, "range") == 0)
    {
      //printf(" Range Mode\n\n");
      mode = RANGE_MODE;
    }
    else
    {
      printf("Invalid mode - valid modes are quick, range, or shmoo\n");
      printf("See --help for more information\n");
      return -3000;
    }
  }
  else
  {
    // default mode - quick
    printf(" Quick Mode\n\n");
    mode = QUICK_MODE;
  }

  if (getCmdLineArgumentString(argc, argv, "kernel", &kernelStr))
  {
    if (strcmp(kernelStr, "kernel1") == 0)
    {
      // printf(" Kernel 1\n\n");
      kernel = KERNEL_MODE;
    }
    else if (strcmp(kernelStr, "kernel2") == 0)
    {
      // printf(" Kernel 2\n\n");
      kernel = KERNEL2_MODE;
    }
    else if (strcmp(kernelStr, "kernel3") == 0)
    {
      // printf(" Kernel 3\n\n");
      kernel = KERNEL3_MODE;
    }
    else
    {
      kernel = DEFAULT;
    }
  }

  if (kernel == KERNEL2_MODE)
  {
  if (checkCmdLineFlag(argc, argv, "bytesperinstruction"))
    {
      bytes_per_inst =  getCmdLineArgumentInt(argc, argv, "bytesperinstruction");
      //printf("Using %d bytes per intruction\n", bytes_per_inst);
    }
    else {
    bytes_per_inst = 4;
    }
    //printf("Bytes_per_instruction: %d\n", bytes_per_inst);
  }

  if (kernel == KERNEL3_MODE) {
      getCmdLineArgumentString(argc, argv, "dtype", &dtypeStr);
  }

  if (checkCmdLineFlag(argc, argv, "htod"))
  {
    htod = true;
  }

  if (checkCmdLineFlag(argc, argv, "dtoh"))
  {
    dtoh = true;
  }

  if (checkCmdLineFlag(argc, argv, "dtod"))
  {
    dtod = true;
  }

#if CUDART_VERSION >= 2020

  if (checkCmdLineFlag(argc, argv, "wc"))
  {
    wc = true;
  }

#endif

  if (checkCmdLineFlag(argc, argv, "cputiming"))
  {
    bDontUseGPUTiming = true;
  }

  if (!htod && !dtoh && !dtod)
  {
    // default:  All
    htod = true;
    dtoh = true;
    dtod = true;
  }

  if (RANGE_MODE == mode)
  {
    if (checkCmdLineFlag(argc, (const char **)argv, "start"))
    {
      start = getCmdLineArgumentInt(argc, argv, "start");

      if (start <= 0)
      {
        printf("Illegal argument - start must be greater than zero\n");
        return -4000;
      }
    }
    else
    {
      printf("Must specify a starting size in range mode\n");
      printf("See --help for more information\n");
      return -5000;
    }

    if (checkCmdLineFlag(argc, (const char **)argv, "end"))
    {
      end = getCmdLineArgumentInt(argc, argv, "end");

      if (end <= 0)
      {
        printf("Illegal argument - end must be greater than zero\n");
        return -6000;
      }

      if (start > end)
      {
        printf("Illegal argument - start is greater than end\n");
        return -7000;
      }
    }
    else
    {
      printf("Must specify an end size in range mode.\n");
      printf("See --help for more information\n");
      return -8000;
    }

    if (checkCmdLineFlag(argc, argv, "increment"))
    {
      increment = getCmdLineArgumentInt(argc, argv, "increment");

      if (increment <= 0)
      {
        printf("Illegal argument - increment must be greater than zero\n");
        return -9000;
      }
    }
    else
    {
      printf("Must specify an increment in user mode\n");
      printf("See --help for more information\n");
      return -10000;
    }
  }

  if (htod)
  {
    testBandwidth((unsigned int)start, (unsigned int)end,
                  (unsigned int)increment, mode, HOST_TO_DEVICE, printmode,
                  memMode, startDevice, endDevice, wc, kernel, bytes_per_inst, dtypeStr);
  }

  if (dtoh)
  {
    testBandwidth((unsigned int)start, (unsigned int)end,
                  (unsigned int)increment, mode, DEVICE_TO_HOST, printmode,
                  memMode, startDevice, endDevice, wc, kernel, bytes_per_inst, dtypeStr);
  }

  if (dtod)
  {
    testBandwidth((unsigned int)start, (unsigned int)end,
                  (unsigned int)increment, mode, DEVICE_TO_DEVICE, printmode,
                  memMode, startDevice, endDevice, wc, kernel, bytes_per_inst, dtypeStr);
  }

  // Ensure that we reset all CUDA Devices in question
  for (int nDevice = startDevice; nDevice <= endDevice; nDevice++)
  {
    hipSetDevice(nDevice);
  }

  return 0;
}

///////////////////////////////////////////////////////////////////////////////
//  Run a bandwidth test
///////////////////////////////////////////////////////////////////////////////

void testBandwidth(unsigned int start, unsigned int end, unsigned int increment,
                   testMode mode, memcpyKind kind, printMode printmode,
                   memoryMode memMode, int startDevice, int endDevice,
                   bool wc, kernelMode kernelMode, int bytes_per_inst, char* dtypeStr)
{
  switch (mode)
  {
  case QUICK_MODE:
    testBandwidthQuick(DEFAULT_SIZE, kind, printmode, memMode, startDevice,
                       endDevice, wc, kernelMode, bytes_per_inst);
    break;

  case RANGE_MODE:
    testBandwidthRange(start, end, increment, kind, printmode, memMode,
                       startDevice, endDevice, wc, kernelMode, bytes_per_inst);
    break;

  case SHMOO_MODE:
        if (strcmp(dtypeStr, "short") == 0)
            testBandwidthShmoo<short>(kind, printmode, memMode, startDevice, endDevice, wc, kernelMode, bytes_per_inst, dtypeStr);
        else if (strcmp(dtypeStr, "int") == 0)
            testBandwidthShmoo<int>(kind, printmode, memMode, startDevice, endDevice, wc, kernelMode, bytes_per_inst, dtypeStr);
        else if (strcmp(dtypeStr, "float") == 0)
            testBandwidthShmoo<float>(kind, printmode, memMode, startDevice, endDevice, wc, kernelMode, bytes_per_inst, dtypeStr);
        else if (strcmp(dtypeStr, "double") == 0)
            testBandwidthShmoo<double>(kind, printmode, memMode, startDevice, endDevice, wc, kernelMode, bytes_per_inst, dtypeStr);
        else
            testBandwidthShmoo<unsigned char>(kind, printmode, memMode, startDevice, endDevice, wc, kernelMode, bytes_per_inst, dtypeStr);
      break;

  default:
    break;
  }
}

//////////////////////////////////////////////////////////////////////
//  Run a quick mode bandwidth test
//////////////////////////////////////////////////////////////////////
void testBandwidthQuick(unsigned int size, memcpyKind kind, printMode printmode,
                        memoryMode memMode, int startDevice, int endDevice,
                        bool wc, kernelMode mode, int bytes_per_inst)
{
  testBandwidthRange(size, size, DEFAULT_INCREMENT, kind, printmode, memMode,
                     startDevice, endDevice, wc, mode, bytes_per_inst);
}

///////////////////////////////////////////////////////////////////////
//  Run a range mode bandwidth test
//////////////////////////////////////////////////////////////////////
void testBandwidthRange(unsigned int start, unsigned int end,
                        unsigned int increment, memcpyKind kind,
                        printMode printmode, memoryMode memMode,
                        int startDevice, int endDevice, bool wc, kernelMode mode, int bytes_per_inst)
{
  // count the number of copies we're going to run
  unsigned int count = 1 + ((end - start) / increment);

  unsigned int *memSizes = (unsigned int *)malloc(count * sizeof(unsigned int));
  double *bandwidths = (double *)malloc(count * sizeof(double));

  // Before calculating the cumulative bandwidth, initialize bandwidths array to
  // NULL
  for (unsigned int i = 0; i < count; i++)
  {
    bandwidths[i] = 0.0;
  }

  // Use the device asked by the user
  for (int currentDevice = startDevice; currentDevice <= endDevice;
       currentDevice++)
  {
    hipSetDevice(currentDevice);

    // run each of the copies
    for (unsigned int i = 0; i < count; i++)
    {
      memSizes[i] = start + i * increment;

      switch (kind)
      {
      case DEVICE_TO_HOST:
        bandwidths[i] += testDeviceToHostTransfer<unsigned char>(memSizes[i], memMode, wc, mode, bytes_per_inst);
        break;

      case HOST_TO_DEVICE:
        bandwidths[i] += testHostToDeviceTransfer<unsigned char>(memSizes[i], memMode, wc, mode, bytes_per_inst);
        break;

      case DEVICE_TO_DEVICE:
        bandwidths[i] += testDeviceToDeviceTransfer<unsigned char>(memSizes[i], mode, bytes_per_inst);
        break;
      }
    }
  } // Complete the bandwidth computation on all the devices

  // print results
  if (printmode == CSV)
  {
    printResultsCSV(memSizes, bandwidths, count, kind, memMode,
                    (1 + endDevice - startDevice), wc, mode, bytes_per_inst, "char");
  }
  else
  {
    printResultsReadable(memSizes, bandwidths, count, kind, memMode,
                         (1 + endDevice - startDevice), wc);
  }

  // clean up
  free(memSizes);
  free(bandwidths);
}

//////////////////////////////////////////////////////////////////////////////
// Intense shmoo mode - covers a large range of values with varying increments
//////////////////////////////////////////////////////////////////////////////
template <typename T>
void testBandwidthShmoo(memcpyKind kind, printMode printmode,
                        memoryMode memMode, int startDevice, int endDevice,
                        bool wc, kernelMode mode, int bytes_per_inst, char* dtypeStr)
{
  // count the number of copies to make
  unsigned int count =
      1 + (SHMOO_LIMIT_20KB / SHMOO_INCREMENT_1KB) +
      ((SHMOO_LIMIT_50KB - SHMOO_LIMIT_20KB) / SHMOO_INCREMENT_2KB) +
      ((SHMOO_LIMIT_100KB - SHMOO_LIMIT_50KB) / SHMOO_INCREMENT_10KB) +
      ((SHMOO_LIMIT_1MB - SHMOO_LIMIT_100KB) / SHMOO_INCREMENT_100KB) +
      ((SHMOO_LIMIT_16MB - SHMOO_LIMIT_1MB) / SHMOO_INCREMENT_1MB) +
      ((SHMOO_LIMIT_32MB - SHMOO_LIMIT_16MB) / SHMOO_INCREMENT_2MB) +
      ((SHMOO_LIMIT_64MB - SHMOO_LIMIT_32MB) / SHMOO_INCREMENT_4MB) +
      ((SHMOO_LIMIT_128MB - SHMOO_LIMIT_64MB) / SHMOO_INCREMENT_8MB) +
      ((SHMOO_LIMIT_256MB - SHMOO_LIMIT_128MB) / SHMOO_INCREMENT_13MB) +
      ((SHMOO_MEMSIZE_MAX - SHMOO_LIMIT_256MB) / SHMOO_INCREMENT_25MB);

  unsigned int *memSizes = (unsigned int *)malloc(count * sizeof(unsigned int));
  double *bandwidths = (double *)malloc(count * sizeof(double));

  // Before calculating the cumulative bandwidth, initialize bandwidths array to
  // NULL
  for (unsigned int i = 0; i < count; i++)
  {
    bandwidths[i] = 0.0;
  }

  // Use the device asked by the user
  for (int currentDevice = startDevice; currentDevice <= endDevice;
       currentDevice++)
  {
    hipSetDevice(currentDevice);
    // Run the shmoo
    int iteration = 0;
    unsigned int memSize = 0;

    while (memSize <= SHMOO_MEMSIZE_MAX)
    {
      if (memSize < SHMOO_LIMIT_20KB)
      {
        memSize += SHMOO_INCREMENT_1KB;
      }
      else if (memSize < SHMOO_LIMIT_50KB)
      {
        memSize += SHMOO_INCREMENT_2KB;
      }
      else if (memSize < SHMOO_LIMIT_100KB)
      {
        memSize += SHMOO_INCREMENT_10KB;
      }
      else if (memSize < SHMOO_LIMIT_1MB)
      {
        memSize += SHMOO_INCREMENT_100KB;
      }
      else if (memSize < SHMOO_LIMIT_16MB)
      {
        memSize += SHMOO_INCREMENT_1MB;
      }
      else if (memSize < SHMOO_LIMIT_32MB)
      {
        memSize += SHMOO_INCREMENT_2MB;
      }
      else if (memSize < SHMOO_LIMIT_64MB)
      {
        memSize += SHMOO_INCREMENT_4MB;
      }
      else if (memSize < SHMOO_LIMIT_128MB)
      {
        memSize += SHMOO_INCREMENT_8MB;
      }
      else if (memSize < SHMOO_LIMIT_256MB)
      {
        memSize += SHMOO_INCREMENT_13MB;
      }
      else
      {
        memSize += SHMOO_INCREMENT_25MB;
      }

      memSizes[iteration] = memSize;

      switch (kind)
      {
      case DEVICE_TO_HOST:
        bandwidths[iteration] +=
            testDeviceToHostTransfer<T>(memSizes[iteration], memMode, wc, mode, bytes_per_inst);
        break;

      case HOST_TO_DEVICE:
        bandwidths[iteration] +=
            testHostToDeviceTransfer<T>(memSizes[iteration], memMode, wc, mode, bytes_per_inst);
        break;

      case DEVICE_TO_DEVICE:
        bandwidths[iteration] +=
            testDeviceToDeviceTransfer<T>(memSizes[iteration], mode, bytes_per_inst);
        break;
      }

      iteration++;
      //printf(".");
      fflush(0);
    }
  } // Complete the bandwidth computation on all the devices

  // print results
  //printf("\n");

  if (CSV == printmode)
  {
    printResultsCSV(memSizes, bandwidths, count, kind, memMode,
                    (1 + endDevice - startDevice), wc, mode, bytes_per_inst, dtypeStr);
  }
  else
  {
    printResultsReadable(memSizes, bandwidths, count, kind, memMode,
                         (1 + endDevice - startDevice), wc);
  }

  // clean up
  free(memSizes);
  free(bandwidths);
}

///////////////////////////////////////////////////////////////////////////////
//  test the bandwidth of a device to host memcopy of a specific size
///////////////////////////////////////////////////////////////////////////////
template <typename T>
float testDeviceToHostTransfer(unsigned int memSize, memoryMode memMode,
                               bool wc, kernelMode mode, int bytes_per_inst)
{
  StopWatchInterface *timer = NULL;
  float elapsedTimeInMs = 0.0f;
  float bandwidthInGBs = 0.0f;
  T *h_idata = NULL;
  T *h_odata = NULL;
  hipEvent_t start, stop;

  sdkCreateTimer(&timer);
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));

  // allocate host memory
  if (PINNED == memMode)
  {
    // pinned memory mode - use special function to get OS-pinned memory
#if CUDART_VERSION >= 2020
    checkCudaErrors(hipHostAlloc((void **)&h_idata, memSize,
                                  (wc) ? hipHostMallocWriteCombined : 0));
    checkCudaErrors(hipHostAlloc((void **)&h_odata, memSize,
                                  (wc) ? hipHostMallocWriteCombined : 0));
#else
    checkCudaErrors(hipHostMalloc((void **)&h_idata, memSize));
    checkCudaErrors(hipHostMalloc((void **)&h_odata, memSize));
#endif
  }
  else
  {
    // pageable memory mode - use malloc
    h_idata = (T *)malloc(memSize);
    h_odata = (T *)malloc(memSize);

    if (h_idata == 0 || h_odata == 0)
    {
      fprintf(stderr, "Not enough memory avaialable on host to run test!\n");
      exit(EXIT_FAILURE);
    }
  }

  // initialize the memory
  for (unsigned int i = 0; i < memSize / sizeof(T); i++)
  {
    h_idata[i] = (T)(i & 0xff);
  }

  // allocate device memory
  T *d_idata;
  checkCudaErrors(hipMalloc((void **)&d_idata, memSize));

  // initialize the device memory
  checkCudaErrors(
      hipMemcpy(d_idata, h_idata, memSize, hipMemcpyHostToDevice));

  // copy data from GPU to Host
  if (PINNED == memMode)
  {
    dim3 grid, block;
    if (bDontUseGPUTiming)
      sdkStartTimer(&timer);
    checkCudaErrors(hipEventRecord(start, 0));
    for (unsigned int i = 0; i < MEMCOPY_ITERATIONS; i++)
    {
       if (KERNEL_MODE == mode)
      {
        // printf("Using new kernel\n");
        // printf("Synchronizing");

        calculateKernelConfig(memSize, grid, block, copyKernel<T>);
        copyKernel<<<grid.x, block.x>>>(d_idata, h_odata, memSize);
      }
      else if (KERNEL2_MODE == mode)
      {
        // printf("Using new kernel");
        calculateKernelConfig(memSize, grid, block, copyKernel2<T>);
        copyKernel2<<<grid.x, block.x>>>(d_idata, h_odata, memSize, bytes_per_inst);
      }
      else if (KERNEL3_MODE == mode)
      {
         MultiplyByTwo multiplyByTwo;

         calculateKernelConfig(memSize, grid, block, transformKernel<T,MultiplyByTwo>);
         //printf("Using new kernel");
         transformKernel<<<grid.x, block.x>>>(h_odata, d_idata, memSize, multiplyByTwo);
      } else {
        // printf("Running origianl");
        checkCudaErrors(hipMemcpyAsync(h_odata, d_idata, memSize,
                                      hipMemcpyDeviceToHost, 0));
      }
    }
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipEventElapsedTime(&elapsedTimeInMs, start, stop));
    if (bDontUseGPUTiming)
    {
      sdkStopTimer(&timer);
      elapsedTimeInMs = sdkGetTimerValue(&timer);
      sdkResetTimer(&timer);
    }
  }
  else
  {
    elapsedTimeInMs = 0;
    for (unsigned int i = 0; i < MEMCOPY_ITERATIONS; i++)
    {
      sdkStartTimer(&timer);
      checkCudaErrors(
          hipMemcpy(h_odata, d_idata, memSize, hipMemcpyDeviceToHost));
      sdkStopTimer(&timer);
      elapsedTimeInMs += sdkGetTimerValue(&timer);
      sdkResetTimer(&timer);
      memset(flush_buf, i, FLUSH_SIZE);
    }
  }

  // calculate bandwidth in GB/s
  double time_s = elapsedTimeInMs / 1e3;
  bandwidthInGBs = (memSize * (float)MEMCOPY_ITERATIONS) / (double)1e9;
  bandwidthInGBs = bandwidthInGBs / time_s;
  // clean up memory
  checkCudaErrors(hipEventDestroy(stop));
  checkCudaErrors(hipEventDestroy(start));
  sdkDeleteTimer(&timer);

  if (PINNED == memMode)
  {
    checkCudaErrors(hipHostFree(h_idata));
    checkCudaErrors(hipHostFree(h_odata));
  }
  else
  {
    free(h_idata);
    free(h_odata);
  }

  checkCudaErrors(hipFree(d_idata));

  return bandwidthInGBs;
}

///////////////////////////////////////////////////////////////////////////////
//! test the bandwidth of a host to device memcopy of a specific size
///////////////////////////////////////////////////////////////////////////////
template <typename T>
float testHostToDeviceTransfer(unsigned int memSize, memoryMode memMode,
                               bool wc, kernelMode mode, int bytes_per_inst)
{
  StopWatchInterface *timer = NULL;
  float elapsedTimeInMs = 0.0f;
  float bandwidthInGBs = 0.0f;
  hipEvent_t start, stop;
  sdkCreateTimer(&timer);
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));

  // allocate host memory
  T *h_odata = NULL;

  if (PINNED == memMode)
  {
#if CUDART_VERSION >= 2020
    // pinned memory mode - use special function to get OS-pinned memory
    checkCudaErrors(hipHostAlloc((void **)&h_odata, memSize,
                                  (wc) ? hipHostMallocWriteCombined : 0));
#else
    // pinned memory mode - use special function to get OS-pinned memory
    checkCudaErrors(hipHostMalloc((void **)&h_odata, memSize));
#endif
  }
  else
  {
    // pageable memory mode - use malloc
    h_odata = (T *)malloc(memSize);

    if (h_odata == 0)
    {
      fprintf(stderr, "Not enough memory available on host to run test!\n");
      exit(EXIT_FAILURE);
    }
  }

  T *h_cacheClear1 = (T *)malloc(CACHE_CLEAR_SIZE);
  T *h_cacheClear2 = (T *)malloc(CACHE_CLEAR_SIZE);

  if (h_cacheClear1 == 0 || h_cacheClear2 == 0)
  {
    fprintf(stderr, "Not enough memory available on host to run test!\n");
    exit(EXIT_FAILURE);
  }

  // initialize the memory
  for (unsigned int i = 0; i < memSize / sizeof(T); i++)
  {
    h_odata[i] = (T)(i & 0xff);
  }

  for (unsigned int i = 0; i < CACHE_CLEAR_SIZE / sizeof(T); i++)
  {
    h_cacheClear1[i] = (T)(i & 0xff);
    h_cacheClear2[i] = (T)(0xff - (i & 0xff));
  }

  // allocate device memory
  T *d_idata;
  checkCudaErrors(hipMalloc((void **)&d_idata, memSize));

  // copy host memory to device memory
  if (PINNED == memMode)
  {
    dim3 grid, block;
    if (bDontUseGPUTiming)
      sdkStartTimer(&timer);
    checkCudaErrors(hipEventRecord(start, 0));
    for (unsigned int i = 0; i < MEMCOPY_ITERATIONS; i++)
    {
      if (KERNEL_MODE == mode)
      {
        // printf("Using new kernel\n");
        // printf("Synchronizing");

        calculateKernelConfig(memSize, grid, block, copyKernel<T>);
        copyKernel<<<grid.x, block.x>>>(h_odata, d_idata, memSize);
      }
      else if (KERNEL2_MODE == mode)
      {
        // printf("Using new kernel");
        calculateKernelConfig(memSize, grid, block, copyKernel2<T>);
        copyKernel2<<<grid.x, block.x>>>(h_odata, d_idata, memSize, bytes_per_inst);
      }
      else if (KERNEL3_MODE == mode)
      {
        MultiplyByTwo multipleByTwo;
        calculateKernelConfig(memSize, grid, block, transformKernel<T, MultiplyByTwo>);
        transformKernel<<<grid.x, block.x>>>(h_odata, d_idata, memSize,multipleByTwo);
      }
      else
      {
	//printf("Using original");
        checkCudaErrors(hipMemcpyAsync(d_idata, h_odata, memSize,
                                        hipMemcpyHostToDevice, 0));
      }
    }
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hipEventElapsedTime(&elapsedTimeInMs, start, stop));
    if (bDontUseGPUTiming)
    {
      sdkStopTimer(&timer);
      elapsedTimeInMs = sdkGetTimerValue(&timer);
      sdkResetTimer(&timer);
    }
  }
  else
  {
    elapsedTimeInMs = 0;
    for (unsigned int i = 0; i < MEMCOPY_ITERATIONS; i++)
    {
      sdkStartTimer(&timer);
      checkCudaErrors(
          hipMemcpy(d_idata, h_odata, memSize, hipMemcpyHostToDevice));
      sdkStopTimer(&timer);
      elapsedTimeInMs += sdkGetTimerValue(&timer);
      sdkResetTimer(&timer);
      memset(flush_buf, i, FLUSH_SIZE);
    }
  }

  // calculate bandwidth in GB/s
  double time_s = elapsedTimeInMs / 1e3;
  bandwidthInGBs = (memSize * (float)MEMCOPY_ITERATIONS) / (double)1e9;
  bandwidthInGBs = bandwidthInGBs / time_s;
  // clean up memory
  checkCudaErrors(hipEventDestroy(stop));
  checkCudaErrors(hipEventDestroy(start));
  sdkDeleteTimer(&timer);

  if (PINNED == memMode)
  {
    checkCudaErrors(hipHostFree(h_odata));
  }
  else
  {
    free(h_odata);
  }

  free(h_cacheClear1);
  free(h_cacheClear2);
  checkCudaErrors(hipFree(d_idata));

  return bandwidthInGBs;
}

///////////////////////////////////////////////////////////////////////////////
//! test the bandwidth of a device to device memcopy of a specific size
///////////////////////////////////////////////////////////////////////////////
template <typename T>
float testDeviceToDeviceTransfer(unsigned int memSize, kernelMode mode, int bytes_per_inst)
{
  StopWatchInterface *timer = NULL;
  float elapsedTimeInMs = 0.0f;
  float bandwidthInGBs = 0.0f;
  hipEvent_t start, stop;

  sdkCreateTimer(&timer);
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));

  // allocate host memory
  T *h_idata = (T *)malloc(memSize);

  if (h_idata == 0)
  {
    fprintf(stderr, "Not enough memory avaialable on host to run test!\n");
    exit(EXIT_FAILURE);
  }

  // initialize the host memory
  for (unsigned int i = 0; i < memSize / sizeof(T); i++)
  {
    h_idata[i] = (T)(i & 0xff);
  }

  // allocate device memory
  T *d_idata;
  checkCudaErrors(hipMalloc((void **)&d_idata, memSize));
  T *d_odata;
  checkCudaErrors(hipMalloc((void **)&d_odata, memSize));
  dim3 grid, block;
  // initialize memory
  if (KERNEL_MODE == mode)
  {
    //

    // printf("Using new kernel\n");
    // printf("Synchronizing");

    calculateKernelConfig(memSize, grid, block, copyKernel<T>);
    copyKernel<<<grid.x, block.x>>>(d_idata, d_odata, memSize);
  }
  else if (KERNEL2_MODE == mode)
  {
    // printf("Using new kernel");
    calculateKernelConfig(memSize, grid, block, copyKernel2<T>);
    copyKernel2<<<1, 1, 1024>>>(d_idata, d_odata, memSize, 4);
  }
  else if (KERNEL3_MODE == mode)
  {
    MultiplyByTwo multipleByTwo;

    calculateKernelConfig(memSize, grid, block, transformKernel<T, MultiplyByTwo>);
    // printf("Using new kernel");
    transformKernel<<<1,1,1024>>>(d_idata, d_odata, memSize,multipleByTwo);
  }
  else
  {
    checkCudaErrors(
        hipMemcpy(d_idata, h_idata, memSize, hipMemcpyHostToDevice));
  }
  // run the memcopy
  sdkStartTimer(&timer);
  checkCudaErrors(hipEventRecord(start, 0));

  for (unsigned int i = 0; i < MEMCOPY_ITERATIONS; i++)
  {
    checkCudaErrors(
        hipMemcpy(d_odata, d_idata, memSize, hipMemcpyDeviceToDevice));
  }

  checkCudaErrors(hipEventRecord(stop, 0));

  // Since device to device memory copies are non-blocking,
  // hipDeviceSynchronize() is required in order to get
  // proper timing.
  checkCudaErrors(hipDeviceSynchronize());

  // get the total elapsed time in ms
  sdkStopTimer(&timer);
  checkCudaErrors(hipEventElapsedTime(&elapsedTimeInMs, start, stop));

  if (bDontUseGPUTiming)
  {
    elapsedTimeInMs = sdkGetTimerValue(&timer);
  }

  // calculate bandwidth in GB/s
  double time_s = elapsedTimeInMs / 1e3;
  bandwidthInGBs = (2.0f * memSize * (float)MEMCOPY_ITERATIONS) / (double)1e9;
  bandwidthInGBs = bandwidthInGBs / time_s;

  // clean up memory
  sdkDeleteTimer(&timer);
  free(h_idata);
  checkCudaErrors(hipEventDestroy(stop));
  checkCudaErrors(hipEventDestroy(start));
  checkCudaErrors(hipFree(d_idata));
  checkCudaErrors(hipFree(d_odata));

  return bandwidthInGBs;
}

/////////////////////////////////////////////////////////
// print results in an easily read format
////////////////////////////////////////////////////////
void printResultsReadable(unsigned int *memSizes, double *bandwidths,
                          unsigned int count, memcpyKind kind,
                          memoryMode memMode, int iNumDevs, bool wc)
{
  printf(" %s Bandwidth, %i Device(s)\n", sMemoryCopyKind[kind], iNumDevs);
  printf(" %s Memory Transfers\n", sMemoryMode[memMode]);

  if (wc)
  {
    printf(" Write-Combined Memory Writes are Enabled");
  }

  printf("   Transfer Size (Bytes)\tBandwidth(GB/s)\n");
  unsigned int i;

  for (i = 0; i < (count - 1); i++)
  {
    printf("   %u\t\t\t%s%.1f\n", memSizes[i],
           (memSizes[i] < 10000) ? "\t" : "", bandwidths[i]);
  }

  printf("   %u\t\t\t%s%.1f\n\n", memSizes[i],
         (memSizes[i] < 10000) ? "\t" : "", bandwidths[i]);
}

///////////////////////////////////////////////////////////////////////////
// print results in a database format
///////////////////////////////////////////////////////////////////////////
void printResultsCSV(unsigned int *memSizes, double *bandwidths,
                     unsigned int count, memcpyKind kind, memoryMode memMode,
                     int iNumDevs, bool wc, kernelMode mode, int bytes_per_inst, std::string dtypeStr)
{
  std::string sConfig;

  // log config information
  if (kind == DEVICE_TO_DEVICE)
  {
    sConfig += "D2D";
  }
  else
  {
    if (kind == DEVICE_TO_HOST)
    {
      sConfig += "D2H";
    }
    else if (kind == HOST_TO_DEVICE)
    {
      sConfig += "H2D";
    }

    if (memMode == PAGEABLE)
    {
      sConfig += "-Paged";
    }
    else if (memMode == PINNED)
    {
      sConfig += "-Pinned";

      if (wc)
      {
        sConfig += "-WriteCombined";
      }
    }
  }
  
	if (mode == DEFAULT)
		sConfig += "-hipMemcpy";
	else if (mode == KERNEL_MODE)
		sConfig += "-kernel1";
	else if (mode == KERNEL2_MODE)
		sConfig += "-kernel2-" + std::to_string(bytes_per_inst) + "BPI";
	else if (mode == KERNEL3_MODE)
		sConfig += "-kernel3-" + dtypeStr;

  unsigned int i;
  double dSeconds = 0.0;

  for (i = 0; i < count; i++)
  {
    dSeconds = (double)memSizes[i] / (bandwidths[i] * (double)(1e9));
    printf(
        "bandwidthTest-%s, Bandwidth = %.1f GB/s, Time = %.5f s, Size = %u "
        "bytes, NumDevsUsed = %d\n",
        sConfig.c_str(), bandwidths[i], dSeconds, memSizes[i], iNumDevs);
  }
}

///////////////////////////////////////////////////////////////////////////
// Print help screen
///////////////////////////////////////////////////////////////////////////
void printHelp(void)
{
  printf("Usage:  bandwidthTest [OPTION]...\n");
  printf(
      "Test the bandwidth for device to host, host to device, and device to "
      "device transfers\n");
  printf("\n");
  printf(
      "Example:  measure the bandwidth of device to host pinned memory copies "
      "in the range 1024 Bytes to 102400 Bytes in 1024 Byte increments\n");
  printf(
      "./bandwidthTest --memory=pinned --mode=range --start=1024 --end=102400 "
      "--increment=1024 --dtoh\n");

  printf("\n");
  printf("Options:\n");
  printf("--help\tDisplay this help menu\n");
  printf("--csv\tPrint results as a CSV\n");
  printf("--device=[deviceno]\tSpecify the device device to be used\n");
  printf("  all - compute cumulative bandwidth on all the devices\n");
  printf("  0,1,2,...,n - Specify any particular device to be used\n");
  printf("--memory=[MEMMODE]\tSpecify which memory mode to use\n");
  printf("  pageable - pageable memory\n");
  printf("  pinned   - non-pageable system memory\n");
  printf("--mode=[MODE]\tSpecify the mode to use\n");
  printf("  quick - performs a quick measurement\n");
  printf("  range - measures a user-specified range of values\n");
  printf("  shmoo - performs an intense shmoo of a large range of values\n");
  printf("   kernel[n] performs quick with new kernels");

  printf("--htod\tMeasure host to device transfers\n");
  printf("--dtoh\tMeasure device to host transfers\n");
  printf("--dtod\tMeasure device to device transfers\n");
#if CUDART_VERSION >= 2020
  printf("--wc\tAllocate pinned memory as write-combined\n");
#endif
  printf("--cputiming\tForce CPU-based timing always\n");

  printf("Range mode options\n");
  printf("--start=[SIZE]\tStarting transfer size in bytes\n");
  printf("--end=[SIZE]\tEnding transfer size in bytes\n");
  printf("--increment=[SIZE]\tIncrement size in bytes\n");
}
