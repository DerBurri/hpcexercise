#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This is a simple test program to measure the memcopy bandwidth of the GPU.
 * It can measure device to device copy bandwidth, host to device copy bandwidth
 * for pageable and pinned memory, and device to host copy bandwidth for
 * pageable and pinned memory.
 *
 * Usage:
 * ./bandwidthTest [option]...
 */

// CUDA runtime
#include <hip/hip_runtime.h>

// includes
#include <hip/hip_runtime_api.h>      // helper functions for CUDA error checking and initialization
#include <helper_functions.h> // helper for shared functions common to CUDA Samples

#include <hip/hip_runtime.h>

#include <cassert>
#include <iostream>
#include <memory>

static const char *sSDKsample = "CUDA Bandwidth Test";

// defines, project
#define MEMCOPY_ITERATIONS 1
#define DEFAULT_SIZE (32 * (1e6))     // 32 M
#define DEFAULT_INCREMENT (4 * (1e6)) // 4 M
#define CACHE_CLEAR_SIZE (16 * (1e6)) // 16 M

// shmoo mode defines
#define SHMOO_MEMSIZE_MAX (64 * (1e6))      // 64 M
#define SHMOO_MEMSIZE_START (1e3)           // 1 KB
#define SHMOO_INCREMENT_1KB (1e3)           // 1 KB
#define SHMOO_INCREMENT_2KB (2 * 1e3)       // 2 KB
#define SHMOO_INCREMENT_10KB (10 * (1e3))   // 10KB
#define SHMOO_INCREMENT_100KB (100 * (1e3)) // 100 KB
#define SHMOO_INCREMENT_1MB (1e6)           // 1 MB
#define SHMOO_INCREMENT_2MB (2 * 1e6)       // 2 MB
#define SHMOO_INCREMENT_4MB (4 * 1e6)       // 4 MB
#define SHMOO_LIMIT_20KB (20 * (1e3))       // 20 KB
#define SHMOO_LIMIT_50KB (50 * (1e3))       // 50 KB
#define SHMOO_LIMIT_100KB (100 * (1e3))     // 100 KB
#define SHMOO_LIMIT_1MB (1e6)               // 1 MB
#define SHMOO_LIMIT_16MB (16 * 1e6)         // 16 MB
#define SHMOO_LIMIT_32MB (32 * 1e6)         // 32 MB

// CPU cache flush
#define FLUSH_SIZE (256 * 1024 * 1024)
char *flush_buf;

// enums, project
enum testMode
{
  QUICK_MODE,
  RANGE_MODE,
  SHMOO_MODE
};
enum memcpyKind
{
  DEVICE_TO_HOST,
  HOST_TO_DEVICE,
  DEVICE_TO_DEVICE
};
enum printMode
{
  USER_READABLE,
  CSV
};
enum memoryMode
{
  PINNED,
  PAGEABLE
};
enum kernelMode
{
  DEFAULT,
  KERNEL_MODE,
  KERNEL2_MODE,
  KERNEL3_MODE
};

__global__ void copyKernel(const unsigned char *in, unsigned char *out, size_t num_bytes);

const char *sMemoryCopyKind[] = {"Device to Host", "Host to Device",
                                 "Device to Device", NULL};

const char *sMemoryMode[] = {"PINNED", "PAGEABLE", NULL};

// if true, use CPU based timing for everything
static bool bDontUseGPUTiming;

int *pArgc = NULL;
char **pArgv = NULL;

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
int runTest(const int argc, const char **argv);
void testBandwidth(unsigned int start, unsigned int end, unsigned int increment,
                   testMode mode, memcpyKind kind, printMode printmode,
                   memoryMode memMode, int startDevice, int endDevice, bool wc, kernelMode kernelMode);
void testBandwidthQuick(unsigned int size, memcpyKind kind, printMode printmode,
                        memoryMode memMode, int startDevice, int endDevice,
                        bool wc, kernelMode mode);
void testBandwidthRange(unsigned int start, unsigned int end,
                        unsigned int increment, memcpyKind kind,
                        printMode printmode, memoryMode memMode,
                        int startDevice, int endDevice, bool wc, kernelMode mode);
void testBandwidthShmoo(memcpyKind kind, printMode printmode,
                        memoryMode memMode, int startDevice, int endDevice,
                        bool wc, kernelMode mode);
float testDeviceToHostTransfer(unsigned int memSize, memoryMode memMode,
                               bool wc, kernelMode mode);
float testHostToDeviceTransfer(unsigned int memSize, memoryMode memMode,
                               bool wc, kernelMode mode);
float testDeviceToDeviceTransfer(unsigned int memSize, kernelMode mode);
void printResultsReadable(unsigned int *memSizes, double *bandwidths,
                          unsigned int count, memcpyKind kind,
                          memoryMode memMode, int iNumDevs, bool wc);
void printResultsCSV(unsigned int *memSizes, double *bandwidths,
                     unsigned int count, memcpyKind kind, memoryMode memMode,
                     int iNumDevs, bool wc);
void printHelp(void);

template <typename T>
void calculateKernelConfig(int numElements, dim3 &grid, dim3 &block, T kernel)
{
  int minGridSize, blockSize;
  hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, kernel, 0, 0);

  grid.x = (numElements + blockSize - 1) / blockSize;
  block.x = blockSize;
  // Calculated Kernel Launch Configuration
  printf("Kernel Launch Configuration ");
  printf("Grid Size: %d, Block Size: %d", grid.x, block.x);
}

////////////////////////////////////////////////////////////////////////////////
// Kernels
////////////////////////////////////////////////////////////////////////////////

__global__ void copyKernel(const unsigned char *in, unsigned char *out, size_t num_bytes)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < num_bytes)
  {
    out[i] = in[i];
  }
}

// __global__ void copyKernel2(unsigned char* in, unsigned char* out, size_t num_bytes, size_t bytes_per_inst) {
//     int i = (blockDim.x * blockIdx.x + threadIdx.x);
//     //int end = num_bytes / bytes_per_inst * bytes_per_inst;
//     if (i < num_bytes  - bytes_per_inst + 1)
//     {
//       printf("copying\n");
//     if (bytes_per_inst == 1) {
//       char* in_vec = reinterpret_cast<char*>(in);
//       char* out_vec = reinterpret_cast<char*>(out);
//       if (i < num_bytes) {
//         out_vec[i / 1] = in_vec[i / 1];
//       }
//     }
//     if (bytes_per_inst == 2) {
//       char2* in_vec = reinterpret_cast<char2*>(in);
//       char2* out_vec = reinterpret_cast<char2*>(out);
//       if (i < num_bytes) {
//         out_vec[i / 2] = in_vec[i / 2];
//       }
//     }
//     if (bytes_per_inst == 4) {
//       char4* in_vec = reinterpret_cast<char4*>(in);
//       char4* out_vec = reinterpret_cast<char4*>(out);
//       if (i < num_bytes) {
//         out_vec[i / 4] = in_vec[i / 4];
//       }
//     }
//     }
// }

__global__ void copyKernel2(const unsigned char *__restrict__ in,
                            unsigned char *__restrict__ out,
                            size_t num_bytes, int bytes_per_inst)
{
  unsigned int total_threads = gridDim.x * blockDim.x;

  // num_bytes = number of all elements to copy
  unsigned int inst_per_thread = num_bytes / bytes_per_inst / total_threads;
  if (inst_per_thread == 0)
    inst_per_thread = 1;
  unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
  printf("thread_id %d, inst_per_thread %d\n", threadIdx.x, inst_per_thread);
  if (num_bytes > idx * (bytes_per_inst * inst_per_thread))
  {
    for (unsigned int element_idx = idx * (bytes_per_inst * inst_per_thread);
         element_idx < (idx + 1) * (bytes_per_inst * inst_per_thread);
         element_idx += bytes_per_inst)
    {
      auto lower_border = idx * (bytes_per_inst * inst_per_thread);
      auto upper_border = (idx + 1) * (bytes_per_inst * inst_per_thread);
      printf("thread_id %d, element_idx: %d, lower_border: %d, upper_border: %d\n", threadIdx.x, element_idx, lower_border, upper_border);
      switch (bytes_per_inst)
      {
      case 4:
        if (element_idx * 4 < num_bytes)
        {
          reinterpret_cast<int *>(out)[idx] =
              reinterpret_cast<const int *>(in)[idx];
        }
        break;
      case 8:
        if (element_idx * 8 < num_bytes)
        {
          reinterpret_cast<int2 *>(out)[idx] =
              reinterpret_cast<const int2 *>(in)[idx];
        }
        break;
      case 16:
        if (element_idx * 16 < num_bytes)
        {
          reinterpret_cast<int4 *>(out)[idx] =
              reinterpret_cast<const int4 *>(in)[idx];
        }
        break;
      default:
        hipError_t error = hipErrorInvalidValue;
        break;
      }
    }
  }
  // Handle remaining bytes if any
  int remaining_bytes = num_bytes % bytes_per_inst;
  int remaining_start_idx = num_bytes - remaining_bytes;

  if (idx == 0 && remaining_bytes > 0)
  {
    for (int i = 0; i < remaining_bytes; ++i)
    {
      out[remaining_start_idx + i] = in[remaining_start_idx + i];
    }
  }
}

template <typename T, class Functor>
__global__ void transformKernel(const T *in, T *out, size_t num_elements, Functor f)
{
  const int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < num_elements)
  {
    T in_value;

    memcpy(&in_value, &in[i], sizeof(T));

    // Apply the functir
    T out_value = f(in_value);

    memcpy(&out[i], &out_value, sizeof(T));
  }
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
  pArgc = &argc;
  pArgv = argv;

  flush_buf = (char *)malloc(FLUSH_SIZE);

  // set logfile name and start logs
  printf("[%s] - Starting...\n", sSDKsample);

  int iRetVal = runTest(argc, (const char **)argv);

  if (iRetVal < 0)
  {
    checkCudaErrors(hipSetDevice(0));
  }

  // finish
  printf("%s\n", (iRetVal == 0) ? "Result = PASS" : "Result = FAIL");

  printf(
      "\nNOTE: The CUDA Samples are not meant for performance measurements. "
      "Results may vary when GPU Boost is enabled.\n");

  free(flush_buf);

  exit((iRetVal == 0) ? EXIT_SUCCESS : EXIT_FAILURE);
}

///////////////////////////////////////////////////////////////////////////////
// Parse args, run the appropriate tests
///////////////////////////////////////////////////////////////////////////////
int runTest(const int argc, const char **argv)
{
  int start = DEFAULT_SIZE;
  int end = DEFAULT_SIZE;
  int startDevice = 0;
  int endDevice = 0;
  int increment = DEFAULT_INCREMENT;
  testMode mode = QUICK_MODE;
  kernelMode kernel = DEFAULT;
  bool htod = false;
  bool dtoh = false;
  bool dtod = false;
  bool wc = false;
  char *modeStr;
  char *kernelStr;
  char *device = NULL;
  printMode printmode = USER_READABLE;
  char *memModeStr = NULL;
  memoryMode memMode = PINNED;

  // process command line args
  if (checkCmdLineFlag(argc, argv, "help"))
  {
    printHelp();
    return 0;
  }

  if (checkCmdLineFlag(argc, argv, "csv"))
  {
    printmode = CSV;
  }

  if (getCmdLineArgumentString(argc, argv, "memory", &memModeStr))
  {
    if (strcmp(memModeStr, "pageable") == 0)
    {
      memMode = PAGEABLE;
    }
    else if (strcmp(memModeStr, "pinned") == 0)
    {
      memMode = PINNED;
    }
    else
    {
      printf("Invalid memory mode - valid modes are pageable or pinned\n");
      printf("See --help for more information\n");
      return -1000;
    }
  }
  else
  {
    // default - pinned memory
    memMode = PINNED;
  }

  if (getCmdLineArgumentString(argc, argv, "device", &device))
  {
    int deviceCount;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);

    if (error_id != hipSuccess)
    {
      printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error_id,
             hipGetErrorString(error_id));
      exit(EXIT_FAILURE);
    }

    if (deviceCount == 0)
    {
      printf("!!!!!No devices found!!!!!\n");
      return -2000;
    }

    if (strcmp(device, "all") == 0)
    {
      printf(
          "\n!!!!!Cumulative Bandwidth to be computed from all the devices "
          "!!!!!!\n\n");
      startDevice = 0;
      endDevice = deviceCount - 1;
    }
    else
    {
      startDevice = endDevice = atoi(device);

      if (startDevice >= deviceCount || startDevice < 0)
      {
        printf(
            "\n!!!!!Invalid GPU number %d given hence default gpu %d will be "
            "used !!!!!\n",
            startDevice, 0);
        startDevice = endDevice = 0;
      }
    }
  }

  printf("Running on...\n\n");

  for (int currentDevice = startDevice; currentDevice <= endDevice;
       currentDevice++)
  {
    hipDeviceProp_t deviceProp;
    hipError_t error_id = hipGetDeviceProperties(&deviceProp, currentDevice);

    if (error_id == hipSuccess)
    {
      printf(" Device %d: %s\n", currentDevice, deviceProp.name);

      if (deviceProp.computeMode == hipComputeModeProhibited)
      {
        fprintf(stderr,
                "Error: device is running in <Compute Mode Prohibited>, no "
                "threads can use ::hipSetDevice().\n");
        checkCudaErrors(hipSetDevice(currentDevice));

        exit(EXIT_FAILURE);
      }
    }
    else
    {
      printf("hipGetDeviceProperties returned %d\n-> %s\n", (int)error_id,
             hipGetErrorString(error_id));
      checkCudaErrors(hipSetDevice(currentDevice));

      exit(EXIT_FAILURE);
    }
  }

  if (getCmdLineArgumentString(argc, argv, "mode", &modeStr))
  {
    // figure out the mode
    if (strcmp(modeStr, "quick") == 0)
    {
      printf(" Quick Mode\n\n");
      mode = QUICK_MODE;
    }
    else if (strcmp(modeStr, "shmoo") == 0)
    {
      printf(" Shmoo Mode\n\n");
      mode = SHMOO_MODE;
    }
    else if (strcmp(modeStr, "range") == 0)
    {
      printf(" Range Mode\n\n");
      mode = RANGE_MODE;
    }
    else
    {
      printf("Invalid mode - valid modes are quick, range, or shmoo\n");
      printf("See --help for more information\n");
      return -3000;
    }
  }
  else
  {
    // default mode - quick
    printf(" Quick Mode\n\n");
    mode = QUICK_MODE;
  }

  if (getCmdLineArgumentString(argc, argv, "kernel", &kernelStr))
  {
    if (strcmp(kernelStr, "kernel1") == 0)
    {
      printf(" Kernel 1\n\n");
      kernel = KERNEL_MODE;
    }
    else if (strcmp(kernelStr, "kernel2") == 0)
    {
      printf(" Kernel 2\n\n");
      kernel = KERNEL2_MODE;
    }
    else
    {
      kernel = DEFAULT;
    }
  }

  if (checkCmdLineFlag(argc, argv, "htod"))
  {
    htod = true;
  }

  if (checkCmdLineFlag(argc, argv, "dtoh"))
  {
    dtoh = true;
  }

  if (checkCmdLineFlag(argc, argv, "dtod"))
  {
    dtod = true;
  }

#if CUDART_VERSION >= 2020

  if (checkCmdLineFlag(argc, argv, "wc"))
  {
    wc = true;
  }

#endif

  if (checkCmdLineFlag(argc, argv, "cputiming"))
  {
    bDontUseGPUTiming = true;
  }

  if (!htod && !dtoh && !dtod)
  {
    // default:  All
    htod = true;
    dtoh = true;
    dtod = true;
  }

  if (RANGE_MODE == mode)
  {
    if (checkCmdLineFlag(argc, (const char **)argv, "start"))
    {
      start = getCmdLineArgumentInt(argc, argv, "start");

      if (start <= 0)
      {
        printf("Illegal argument - start must be greater than zero\n");
        return -4000;
      }
    }
    else
    {
      printf("Must specify a starting size in range mode\n");
      printf("See --help for more information\n");
      return -5000;
    }

    if (checkCmdLineFlag(argc, (const char **)argv, "end"))
    {
      end = getCmdLineArgumentInt(argc, argv, "end");

      if (end <= 0)
      {
        printf("Illegal argument - end must be greater than zero\n");
        return -6000;
      }

      if (start > end)
      {
        printf("Illegal argument - start is greater than end\n");
        return -7000;
      }
    }
    else
    {
      printf("Must specify an end size in range mode.\n");
      printf("See --help for more information\n");
      return -8000;
    }

    if (checkCmdLineFlag(argc, argv, "increment"))
    {
      increment = getCmdLineArgumentInt(argc, argv, "increment");

      if (increment <= 0)
      {
        printf("Illegal argument - increment must be greater than zero\n");
        return -9000;
      }
    }
    else
    {
      printf("Must specify an increment in user mode\n");
      printf("See --help for more information\n");
      return -10000;
    }
  }

  if (htod)
  {
    testBandwidth((unsigned int)start, (unsigned int)end,
                  (unsigned int)increment, mode, HOST_TO_DEVICE, printmode,
                  memMode, startDevice, endDevice, wc, kernel);
  }

  if (dtoh)
  {
    testBandwidth((unsigned int)start, (unsigned int)end,
                  (unsigned int)increment, mode, DEVICE_TO_HOST, printmode,
                  memMode, startDevice, endDevice, wc, kernel);
  }

  if (dtod)
  {
    testBandwidth((unsigned int)start, (unsigned int)end,
                  (unsigned int)increment, mode, DEVICE_TO_DEVICE, printmode,
                  memMode, startDevice, endDevice, wc, kernel);
  }

  // Ensure that we reset all CUDA Devices in question
  for (int nDevice = startDevice; nDevice <= endDevice; nDevice++)
  {
    hipSetDevice(nDevice);
  }

  return 0;
}

///////////////////////////////////////////////////////////////////////////////
//  Run a bandwidth test
///////////////////////////////////////////////////////////////////////////////
void testBandwidth(unsigned int start, unsigned int end, unsigned int increment,
                   testMode mode, memcpyKind kind, printMode printmode,
                   memoryMode memMode, int startDevice, int endDevice,
                   bool wc, kernelMode kernelMode)
{
  switch (mode)
  {
  case QUICK_MODE:
    testBandwidthQuick(DEFAULT_SIZE, kind, printmode, memMode, startDevice,
                       endDevice, wc, kernelMode);
    break;

  case RANGE_MODE:
    testBandwidthRange(start, end, increment, kind, printmode, memMode,
                       startDevice, endDevice, wc, kernelMode);
    break;

  case SHMOO_MODE:
    testBandwidthShmoo(kind, printmode, memMode, startDevice, endDevice, wc, kernelMode);
    break;

  default:
    break;
  }
}

//////////////////////////////////////////////////////////////////////
//  Run a quick mode bandwidth test
//////////////////////////////////////////////////////////////////////
void testBandwidthQuick(unsigned int size, memcpyKind kind, printMode printmode,
                        memoryMode memMode, int startDevice, int endDevice,
                        bool wc, kernelMode mode)
{
  testBandwidthRange(size, size, DEFAULT_INCREMENT, kind, printmode, memMode,
                     startDevice, endDevice, wc, mode);
}

///////////////////////////////////////////////////////////////////////
//  Run a range mode bandwidth test
//////////////////////////////////////////////////////////////////////
void testBandwidthRange(unsigned int start, unsigned int end,
                        unsigned int increment, memcpyKind kind,
                        printMode printmode, memoryMode memMode,
                        int startDevice, int endDevice, bool wc, kernelMode mode)
{
  // count the number of copies we're going to run
  unsigned int count = 1 + ((end - start) / increment);

  unsigned int *memSizes = (unsigned int *)malloc(count * sizeof(unsigned int));
  double *bandwidths = (double *)malloc(count * sizeof(double));

  // Before calculating the cumulative bandwidth, initialize bandwidths array to
  // NULL
  for (unsigned int i = 0; i < count; i++)
  {
    bandwidths[i] = 0.0;
  }

  // Use the device asked by the user
  for (int currentDevice = startDevice; currentDevice <= endDevice;
       currentDevice++)
  {
    hipSetDevice(currentDevice);

    // run each of the copies
    for (unsigned int i = 0; i < count; i++)
    {
      memSizes[i] = start + i * increment;

      switch (kind)
      {
      case DEVICE_TO_HOST:
        bandwidths[i] += testDeviceToHostTransfer(memSizes[i], memMode, wc, mode);
        break;

      case HOST_TO_DEVICE:
        bandwidths[i] += testHostToDeviceTransfer(memSizes[i], memMode, wc, mode);
        break;

      case DEVICE_TO_DEVICE:
        bandwidths[i] += testDeviceToDeviceTransfer(memSizes[i], mode);
        break;
      }
    }
  } // Complete the bandwidth computation on all the devices

  // print results
  if (printmode == CSV)
  {
    printResultsCSV(memSizes, bandwidths, count, kind, memMode,
                    (1 + endDevice - startDevice), wc);
  }
  else
  {
    printResultsReadable(memSizes, bandwidths, count, kind, memMode,
                         (1 + endDevice - startDevice), wc);
  }

  // clean up
  free(memSizes);
  free(bandwidths);
}

//////////////////////////////////////////////////////////////////////////////
// Intense shmoo mode - covers a large range of values with varying increments
//////////////////////////////////////////////////////////////////////////////
void testBandwidthShmoo(memcpyKind kind, printMode printmode,
                        memoryMode memMode, int startDevice, int endDevice,
                        bool wc, kernelMode mode)
{
  // count the number of copies to make
  unsigned int count =
      1 + (SHMOO_LIMIT_20KB / SHMOO_INCREMENT_1KB) +
      ((SHMOO_LIMIT_50KB - SHMOO_LIMIT_20KB) / SHMOO_INCREMENT_2KB) +
      ((SHMOO_LIMIT_100KB - SHMOO_LIMIT_50KB) / SHMOO_INCREMENT_10KB) +
      ((SHMOO_LIMIT_1MB - SHMOO_LIMIT_100KB) / SHMOO_INCREMENT_100KB) +
      ((SHMOO_LIMIT_16MB - SHMOO_LIMIT_1MB) / SHMOO_INCREMENT_1MB) +
      ((SHMOO_LIMIT_32MB - SHMOO_LIMIT_16MB) / SHMOO_INCREMENT_2MB) +
      ((SHMOO_MEMSIZE_MAX - SHMOO_LIMIT_32MB) / SHMOO_INCREMENT_4MB);

  unsigned int *memSizes = (unsigned int *)malloc(count * sizeof(unsigned int));
  double *bandwidths = (double *)malloc(count * sizeof(double));

  // Before calculating the cumulative bandwidth, initialize bandwidths array to
  // NULL
  for (unsigned int i = 0; i < count; i++)
  {
    bandwidths[i] = 0.0;
  }

  // Use the device asked by the user
  for (int currentDevice = startDevice; currentDevice <= endDevice;
       currentDevice++)
  {
    hipSetDevice(currentDevice);
    // Run the shmoo
    int iteration = 0;
    unsigned int memSize = 0;

    while (memSize <= SHMOO_MEMSIZE_MAX)
    {
      if (memSize < SHMOO_LIMIT_20KB)
      {
        memSize += SHMOO_INCREMENT_1KB;
      }
      else if (memSize < SHMOO_LIMIT_50KB)
      {
        memSize += SHMOO_INCREMENT_2KB;
      }
      else if (memSize < SHMOO_LIMIT_100KB)
      {
        memSize += SHMOO_INCREMENT_10KB;
      }
      else if (memSize < SHMOO_LIMIT_1MB)
      {
        memSize += SHMOO_INCREMENT_100KB;
      }
      else if (memSize < SHMOO_LIMIT_16MB)
      {
        memSize += SHMOO_INCREMENT_1MB;
      }
      else if (memSize < SHMOO_LIMIT_32MB)
      {
        memSize += SHMOO_INCREMENT_2MB;
      }
      else
      {
        memSize += SHMOO_INCREMENT_4MB;
      }

      memSizes[iteration] = memSize;

      switch (kind)
      {
      case DEVICE_TO_HOST:
        bandwidths[iteration] +=
            testDeviceToHostTransfer(memSizes[iteration], memMode, wc, mode);
        break;

      case HOST_TO_DEVICE:
        bandwidths[iteration] +=
            testHostToDeviceTransfer(memSizes[iteration], memMode, wc, mode);
        break;

      case DEVICE_TO_DEVICE:
        bandwidths[iteration] +=
            testDeviceToDeviceTransfer(memSizes[iteration], mode);
        break;
      }

      iteration++;
      printf(".");
      fflush(0);
    }
  } // Complete the bandwidth computation on all the devices

  // print results
  printf("\n");

  if (CSV == printmode)
  {
    printResultsCSV(memSizes, bandwidths, count, kind, memMode,
                    (1 + endDevice - startDevice), wc);
  }
  else
  {
    printResultsReadable(memSizes, bandwidths, count, kind, memMode,
                         (1 + endDevice - startDevice), wc);
  }

  // clean up
  free(memSizes);
  free(bandwidths);
}

///////////////////////////////////////////////////////////////////////////////
//  test the bandwidth of a device to host memcopy of a specific size
///////////////////////////////////////////////////////////////////////////////
float testDeviceToHostTransfer(unsigned int memSize, memoryMode memMode,
                               bool wc, kernelMode mode)
{
  StopWatchInterface *timer = NULL;
  float elapsedTimeInMs = 0.0f;
  float bandwidthInGBs = 0.0f;
  unsigned char *h_idata = NULL;
  unsigned char *h_odata = NULL;
  hipEvent_t start, stop;

  sdkCreateTimer(&timer);
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));

  // allocate host memory
  if (PINNED == memMode)
  {
    // pinned memory mode - use special function to get OS-pinned memory
#if CUDART_VERSION >= 2020
    checkCudaErrors(hipHostAlloc((void **)&h_idata, memSize,
                                  (wc) ? hipHostMallocWriteCombined : 0));
    checkCudaErrors(hipHostAlloc((void **)&h_odata, memSize,
                                  (wc) ? hipHostMallocWriteCombined : 0));
#else
    checkCudaErrors(hipHostMalloc((void **)&h_idata, memSize));
    checkCudaErrors(hipHostMalloc((void **)&h_odata, memSize));
#endif
  }
  else
  {
    // pageable memory mode - use malloc
    h_idata = (unsigned char *)malloc(memSize);
    h_odata = (unsigned char *)malloc(memSize);

    if (h_idata == 0 || h_odata == 0)
    {
      fprintf(stderr, "Not enough memory avaialable on host to run test!\n");
      exit(EXIT_FAILURE);
    }
  }

  // initialize the memory
  for (unsigned int i = 0; i < memSize / sizeof(unsigned char); i++)
  {
    h_idata[i] = (unsigned char)(i & 0xff);
  }

  // allocate device memory
  unsigned char *d_idata;
  checkCudaErrors(hipMalloc((void **)&d_idata, memSize));

  // initialize the device memory
  checkCudaErrors(
      hipMemcpy(d_idata, h_idata, memSize, hipMemcpyHostToDevice));

  // copy data from GPU to Host
  if (PINNED == memMode)
  {
    if (bDontUseGPUTiming)
      sdkStartTimer(&timer);
    checkCudaErrors(hipEventRecord(start, 0));
    for (unsigned int i = 0; i < MEMCOPY_ITERATIONS; i++)
    {
      checkCudaErrors(hipMemcpyAsync(h_odata, d_idata, memSize,
                                      hipMemcpyDeviceToHost, 0));
    }
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipEventElapsedTime(&elapsedTimeInMs, start, stop));
    if (bDontUseGPUTiming)
    {
      sdkStopTimer(&timer);
      elapsedTimeInMs = sdkGetTimerValue(&timer);
      sdkResetTimer(&timer);
    }
  }
  else
  {
    elapsedTimeInMs = 0;
    for (unsigned int i = 0; i < MEMCOPY_ITERATIONS; i++)
    {
      sdkStartTimer(&timer);
      checkCudaErrors(
          hipMemcpy(h_odata, d_idata, memSize, hipMemcpyDeviceToHost));
      sdkStopTimer(&timer);
      elapsedTimeInMs += sdkGetTimerValue(&timer);
      sdkResetTimer(&timer);
      memset(flush_buf, i, FLUSH_SIZE);
    }
  }

  // calculate bandwidth in GB/s
  double time_s = elapsedTimeInMs / 1e3;
  bandwidthInGBs = (memSize * (float)MEMCOPY_ITERATIONS) / (double)1e9;
  bandwidthInGBs = bandwidthInGBs / time_s;
  // clean up memory
  checkCudaErrors(hipEventDestroy(stop));
  checkCudaErrors(hipEventDestroy(start));
  sdkDeleteTimer(&timer);

  if (PINNED == memMode)
  {
    checkCudaErrors(hipHostFree(h_idata));
    checkCudaErrors(hipHostFree(h_odata));
  }
  else
  {
    free(h_idata);
    free(h_odata);
  }

  checkCudaErrors(hipFree(d_idata));

  return bandwidthInGBs;
}

///////////////////////////////////////////////////////////////////////////////
//! test the bandwidth of a host to device memcopy of a specific size
///////////////////////////////////////////////////////////////////////////////
float testHostToDeviceTransfer(unsigned int memSize, memoryMode memMode,
                               bool wc, kernelMode mode)
{
  StopWatchInterface *timer = NULL;
  float elapsedTimeInMs = 0.0f;
  float bandwidthInGBs = 0.0f;
  hipEvent_t start, stop;
  sdkCreateTimer(&timer);
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));

  // allocate host memory
  unsigned char *h_odata = NULL;

  if (PINNED == memMode)
  {
#if CUDART_VERSION >= 2020
    // pinned memory mode - use special function to get OS-pinned memory
    checkCudaErrors(hipHostAlloc((void **)&h_odata, memSize,
                                  (wc) ? hipHostMallocWriteCombined : 0));
#else
    // pinned memory mode - use special function to get OS-pinned memory
    checkCudaErrors(hipHostMalloc((void **)&h_odata, memSize));
#endif
  }
  else
  {
    // pageable memory mode - use malloc
    h_odata = (unsigned char *)malloc(memSize);

    if (h_odata == 0)
    {
      fprintf(stderr, "Not enough memory available on host to run test!\n");
      exit(EXIT_FAILURE);
    }
  }

  unsigned char *h_cacheClear1 = (unsigned char *)malloc(CACHE_CLEAR_SIZE);
  unsigned char *h_cacheClear2 = (unsigned char *)malloc(CACHE_CLEAR_SIZE);

  if (h_cacheClear1 == 0 || h_cacheClear2 == 0)
  {
    fprintf(stderr, "Not enough memory available on host to run test!\n");
    exit(EXIT_FAILURE);
  }

  // initialize the memory
  for (unsigned int i = 0; i < memSize / sizeof(unsigned char); i++)
  {
    h_odata[i] = (unsigned char)(i & 0xff);
  }

  for (unsigned int i = 0; i < CACHE_CLEAR_SIZE / sizeof(unsigned char); i++)
  {
    h_cacheClear1[i] = (unsigned char)(i & 0xff);
    h_cacheClear2[i] = (unsigned char)(0xff - (i & 0xff));
  }

  // allocate device memory
  unsigned char *d_idata;
  checkCudaErrors(hipMalloc((void **)&d_idata, memSize));

  // copy host memory to device memory
  if (PINNED == memMode)
  {
    dim3 grid, block;
    if (bDontUseGPUTiming)
      sdkStartTimer(&timer);
    checkCudaErrors(hipEventRecord(start, 0));
    for (unsigned int i = 0; i < MEMCOPY_ITERATIONS; i++)
    {
      if (KERNEL_MODE == mode)
      {
        // printf("Using new kernel\n");
        // printf("Synchronizing");

        calculateKernelConfig(memSize, grid, block, copyKernel);
        copyKernel<<<grid.x, block.x>>>(h_odata, d_idata, memSize);
      }
      else if (KERNEL2_MODE == mode)
      {
        // printf("Using new kernel");
        int bytes_per_inst = 8;
        calculateKernelConfig(memSize, grid, block, copyKernel2);
        copyKernel2<<<grid.x, block.x>>>(h_odata, d_idata, memSize, bytes_per_inst);
      }
      else if (KERNEL3_MODE == mode)
      {
        // calculateKernelConfig(memSize, grid, block, transformKernel);
        // printf("Using new kernel");
        // transformKernel<<<1,1,1024>>>(h_odata, d_idata, memSize, [=] __device__ (auto x) { return 0 * 0.5f;});
      }
      else
      {
        printf("Using original");
        checkCudaErrors(hipMemcpyAsync(d_idata, h_odata, memSize,
                                        hipMemcpyHostToDevice, 0));
      }
    }
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hipEventElapsedTime(&elapsedTimeInMs, start, stop));
    if (bDontUseGPUTiming)
    {
      sdkStopTimer(&timer);
      elapsedTimeInMs = sdkGetTimerValue(&timer);
      sdkResetTimer(&timer);
    }
  }
  else
  {
    elapsedTimeInMs = 0;
    for (unsigned int i = 0; i < MEMCOPY_ITERATIONS; i++)
    {
      sdkStartTimer(&timer);
      checkCudaErrors(
          hipMemcpy(d_idata, h_odata, memSize, hipMemcpyHostToDevice));
      sdkStopTimer(&timer);
      elapsedTimeInMs += sdkGetTimerValue(&timer);
      sdkResetTimer(&timer);
      memset(flush_buf, i, FLUSH_SIZE);
    }
  }

  // calculate bandwidth in GB/s
  double time_s = elapsedTimeInMs / 1e3;
  bandwidthInGBs = (memSize * (float)MEMCOPY_ITERATIONS) / (double)1e9;
  bandwidthInGBs = bandwidthInGBs / time_s;
  // clean up memory
  checkCudaErrors(hipEventDestroy(stop));
  checkCudaErrors(hipEventDestroy(start));
  sdkDeleteTimer(&timer);

  if (PINNED == memMode)
  {
    checkCudaErrors(hipHostFree(h_odata));
  }
  else
  {
    free(h_odata);
  }

  free(h_cacheClear1);
  free(h_cacheClear2);
  checkCudaErrors(hipFree(d_idata));

  return bandwidthInGBs;
}

///////////////////////////////////////////////////////////////////////////////
//! test the bandwidth of a device to device memcopy of a specific size
///////////////////////////////////////////////////////////////////////////////
float testDeviceToDeviceTransfer(unsigned int memSize, kernelMode mode)
{
  StopWatchInterface *timer = NULL;
  float elapsedTimeInMs = 0.0f;
  float bandwidthInGBs = 0.0f;
  hipEvent_t start, stop;

  sdkCreateTimer(&timer);
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));

  // allocate host memory
  unsigned char *h_idata = (unsigned char *)malloc(memSize);

  if (h_idata == 0)
  {
    fprintf(stderr, "Not enough memory avaialable on host to run test!\n");
    exit(EXIT_FAILURE);
  }

  // initialize the host memory
  for (unsigned int i = 0; i < memSize / sizeof(unsigned char); i++)
  {
    h_idata[i] = (unsigned char)(i & 0xff);
  }

  // allocate device memory
  unsigned char *d_idata;
  checkCudaErrors(hipMalloc((void **)&d_idata, memSize));
  unsigned char *d_odata;
  checkCudaErrors(hipMalloc((void **)&d_odata, memSize));
  dim3 grid, block;
  // initialize memory
  if (KERNEL_MODE == mode)
  {
    //

    // printf("Using new kernel\n");
    // printf("Synchronizing");

    calculateKernelConfig(memSize, grid, block, copyKernel);
    copyKernel<<<grid.x, block.x>>>(d_idata, d_odata, memSize);
  }
  else if (KERNEL2_MODE == mode)
  {
    // printf("Using new kernel");
    calculateKernelConfig(memSize, grid, block, copyKernel2);
    copyKernel2<<<1, 1, 1024>>>(d_idata, d_odata, memSize, 4);
  }
  else if (KERNEL3_MODE == mode)
  {
    // calculateKernelConfig(memSize, grid, block, transformKernel);
    // printf("Using new kernel");
    // transformKernel<<<1,1,1024>>>(h_odata, d_idata, memSize, [=] __device__ (auto x) { return 0 * 0.5f;});
  }
  else
  {
    checkCudaErrors(
        hipMemcpy(d_idata, h_idata, memSize, hipMemcpyHostToDevice));
  }
  // run the memcopy
  sdkStartTimer(&timer);
  checkCudaErrors(hipEventRecord(start, 0));

  for (unsigned int i = 0; i < MEMCOPY_ITERATIONS; i++)
  {
    checkCudaErrors(
        hipMemcpy(d_odata, d_idata, memSize, hipMemcpyDeviceToDevice));
  }

  checkCudaErrors(hipEventRecord(stop, 0));

  // Since device to device memory copies are non-blocking,
  // hipDeviceSynchronize() is required in order to get
  // proper timing.
  checkCudaErrors(hipDeviceSynchronize());

  // get the total elapsed time in ms
  sdkStopTimer(&timer);
  checkCudaErrors(hipEventElapsedTime(&elapsedTimeInMs, start, stop));

  if (bDontUseGPUTiming)
  {
    elapsedTimeInMs = sdkGetTimerValue(&timer);
  }

  // calculate bandwidth in GB/s
  double time_s = elapsedTimeInMs / 1e3;
  bandwidthInGBs = (2.0f * memSize * (float)MEMCOPY_ITERATIONS) / (double)1e9;
  bandwidthInGBs = bandwidthInGBs / time_s;

  // clean up memory
  sdkDeleteTimer(&timer);
  free(h_idata);
  checkCudaErrors(hipEventDestroy(stop));
  checkCudaErrors(hipEventDestroy(start));
  checkCudaErrors(hipFree(d_idata));
  checkCudaErrors(hipFree(d_odata));

  return bandwidthInGBs;
}

/////////////////////////////////////////////////////////
// print results in an easily read format
////////////////////////////////////////////////////////
void printResultsReadable(unsigned int *memSizes, double *bandwidths,
                          unsigned int count, memcpyKind kind,
                          memoryMode memMode, int iNumDevs, bool wc)
{
  printf(" %s Bandwidth, %i Device(s)\n", sMemoryCopyKind[kind], iNumDevs);
  printf(" %s Memory Transfers\n", sMemoryMode[memMode]);

  if (wc)
  {
    printf(" Write-Combined Memory Writes are Enabled");
  }

  printf("   Transfer Size (Bytes)\tBandwidth(GB/s)\n");
  unsigned int i;

  for (i = 0; i < (count - 1); i++)
  {
    printf("   %u\t\t\t%s%.1f\n", memSizes[i],
           (memSizes[i] < 10000) ? "\t" : "", bandwidths[i]);
  }

  printf("   %u\t\t\t%s%.1f\n\n", memSizes[i],
         (memSizes[i] < 10000) ? "\t" : "", bandwidths[i]);
}

///////////////////////////////////////////////////////////////////////////
// print results in a database format
///////////////////////////////////////////////////////////////////////////
void printResultsCSV(unsigned int *memSizes, double *bandwidths,
                     unsigned int count, memcpyKind kind, memoryMode memMode,
                     int iNumDevs, bool wc)
{
  std::string sConfig;

  // log config information
  if (kind == DEVICE_TO_DEVICE)
  {
    sConfig += "D2D";
  }
  else
  {
    if (kind == DEVICE_TO_HOST)
    {
      sConfig += "D2H";
    }
    else if (kind == HOST_TO_DEVICE)
    {
      sConfig += "H2D";
    }

    if (memMode == PAGEABLE)
    {
      sConfig += "-Paged";
    }
    else if (memMode == PINNED)
    {
      sConfig += "-Pinned";

      if (wc)
      {
        sConfig += "-WriteCombined";
      }
    }
  }

  unsigned int i;
  double dSeconds = 0.0;

  for (i = 0; i < count; i++)
  {
    dSeconds = (double)memSizes[i] / (bandwidths[i] * (double)(1e9));
    printf(
        "bandwidthTest-%s, Bandwidth = %.1f GB/s, Time = %.5f s, Size = %u "
        "bytes, NumDevsUsed = %d\n",
        sConfig.c_str(), bandwidths[i], dSeconds, memSizes[i], iNumDevs);
  }
}

///////////////////////////////////////////////////////////////////////////
// Print help screen
///////////////////////////////////////////////////////////////////////////
void printHelp(void)
{
  printf("Usage:  bandwidthTest [OPTION]...\n");
  printf(
      "Test the bandwidth for device to host, host to device, and device to "
      "device transfers\n");
  printf("\n");
  printf(
      "Example:  measure the bandwidth of device to host pinned memory copies "
      "in the range 1024 Bytes to 102400 Bytes in 1024 Byte increments\n");
  printf(
      "./bandwidthTest --memory=pinned --mode=range --start=1024 --end=102400 "
      "--increment=1024 --dtoh\n");

  printf("\n");
  printf("Options:\n");
  printf("--help\tDisplay this help menu\n");
  printf("--csv\tPrint results as a CSV\n");
  printf("--device=[deviceno]\tSpecify the device device to be used\n");
  printf("  all - compute cumulative bandwidth on all the devices\n");
  printf("  0,1,2,...,n - Specify any particular device to be used\n");
  printf("--memory=[MEMMODE]\tSpecify which memory mode to use\n");
  printf("  pageable - pageable memory\n");
  printf("  pinned   - non-pageable system memory\n");
  printf("--mode=[MODE]\tSpecify the mode to use\n");
  printf("  quick - performs a quick measurement\n");
  printf("  range - measures a user-specified range of values\n");
  printf("  shmoo - performs an intense shmoo of a large range of values\n");
  printf("   kernel[n] performs quick with new kernels");

  printf("--htod\tMeasure host to device transfers\n");
  printf("--dtoh\tMeasure device to host transfers\n");
  printf("--dtod\tMeasure device to device transfers\n");
#if CUDART_VERSION >= 2020
  printf("--wc\tAllocate pinned memory as write-combined\n");
#endif
  printf("--cputiming\tForce CPU-based timing always\n");

  printf("Range mode options\n");
  printf("--start=[SIZE]\tStarting transfer size in bytes\n");
  printf("--end=[SIZE]\tEnding transfer size in bytes\n");
  printf("--increment=[SIZE]\tIncrement size in bytes\n");
}
